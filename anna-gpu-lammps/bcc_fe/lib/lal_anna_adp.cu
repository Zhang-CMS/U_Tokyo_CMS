#include "hip/hip_runtime.h"
//* Device code *---------------------------------------
//      Physics-informed Neural Network Potential
//             Accelerated by GPU
//______________________________________________________
//  begin:  Monday August 07, 2023
//  email:  meng_zhang@metall.t.u-tokyo.ac.jp
//          junya_inoue@metall.t.u-tokyo.ac.jp
//______________________________________________________
//------------------------------------------------------

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( rho_tex,float);
_texture( imu_tex,float);
_texture( lambda_tex,float);
_texture( ladp_tex,float);
_texture( weight_tex,float);
_texture( bias_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( rho_tex, int2);
_texture( imu_tex, int2);
_texture( lambda_tex,int2);
_texture( ladp_tex,int2);
_texture( weight_tex,int2);
_texture( bias_tex,int2);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define weight_tex weight_all;
#define bias_tex bias_all
#define rho_tex adp_rho
#define imu_tex adp_mu
#define lambda_tex adp_lambda
#define ladp_tex ladp_params
#endif

#else
#define pos_tex x_
#define rho_tex adp_rho
#define imu_tex adp_mu
#define lambda_tex adp_lambda
#define ladp_tex ladp_params
#define weight_tex weight_all;
#define bias_tex bias_all
#endif

#define MY_PI (numtyp)3.14159265358979323846
#define coeff_a (numtyp)1.7
#define coeff_b (numtyp)0.6

#if (SHUFFLE_AVAIL == 0)

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(dev_nbor, in_out, ii, nbor_pitch, num_in,                  \
                 num_ou, tid, t_per_atom, offset)                           \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 20] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if (offset < s) {                                               \
                red_accj_in[tid] += red_accj_in[tid + s];                   \
            }                                                               \
        }                                                                   \
        num_in = red_accj_in[tid];                                          \
    } else {                                                                \
        in_out[offset] = num_in;                                            \
        in_out[offset + 20] = num_ou;                                       \
    }                                                                       \
    if(offset == 0) {                                                       \
        dev_nbor[ii + nbor_pitch] = num_in;                                 \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accGi[BLOCK_PAIR];

#define acc_Gi(G_i, nsf, tid, t_per_atom, offset)                           \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accGi[tid] = G_i[i]                                         \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s)                                             \
                    red_accGi[tid] += red_accGi[tid + s];                   \
            }                                                               \
            G_i[i] = red_accG[tid - offset];                                \
        }                                                                   \
    }

#define local_allocate_acc_hide()                                           \
    __local numtyp red_acchid[BLOCK_PAIR];

#define acc_hide(hid, t_hid, nnod, tid, t_per_atom, offset)                 \
    if(t_per_atom > 1) {                                                    \
        for(int i = 0; i < nnod; i++) {                                     \
            red_acchid[tid] = t_hid[i];                                     \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s)                                             \
                    red_acchid[tid] += red_acchid[tid + s];                 \
            }                                                               \
            hid[i] = red_acchid[tid - offset];                              \
        }                                                                   \
    }

#define local_allocate_store_energy_padp()                                  \
    __local acctyp red_accrho[BLOCK_PAIR];                                  \
    __local acctyp red_accmu[BLOCK_PAIR][3];                                \
    __local acctyp red_acclamb[BLOCK_PAIR][6];

#define store_energy_padp(rho_i, mu_i, lambda_i, c1F, c2F, adp_rho,         \
                          adp_mu, adp_lambda, energy, ii, i, nall, inum,    \
                          tid, t_per_atom, offset, eflag, vflag, engv)      \
    if (t_per_atom > 1) {                                                   \
        red_accrho[tid] = rho_i;                                            \
         for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {            \
            simdsync();                                                     \
            if (offset < s)                                                 \
                red_accrho[tid] += red_accrho[tid + s];                     \
        }                                                                   \
        rho_i = red_accrho[tid];                                            \
        for (int k = 0; k < 3; k++) {                                       \
            red_accmu[tid][k] = mu_i[k];                                    \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s)                                             \
                    red_accmu[tid][k] += red_accmu[tid + s][k];             \
           }                                                                \
           mu_i[k] = red_accmu[tid][k];                                     \
        }                                                                   \
        for (int k = 0; k < 6; k++) {                                       \
            red_acclamb[tid][k] = mu_i[k];                                  \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s)                                             \
                    red_acclamb[tid][k] += red_acclamb[tid + s][k];         \
           }                                                                \
           lambda_i[k] = red_acclamb[tid][k];                               \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        acctyp v_i = lambda_i[0] + lambda_i[1] + lambda_i[2];               \
        acctyp sum_mu = 0.0, sum_lamb = 0.0;                                \
        adp_rho[i] = rho_i;                                                 \
        for(int k = 0; k < 6; k++) {                                        \
            if(k < 3) {                                                     \
                sum_mu += mu_i[k] * mu_i[k];                                \
                sum_lamb += lambda_i[k] * lambda_i[k];                      \
                adp_mu[i + k * nall] = mu_i[k];                             \
            }                                                               \
            adp_lambda[i + k * nall] = lambda_i[k];                         \
        }                                                                   \
        sum_lamb += 2.0 * (pow(lambda_i[3], 2) + pow(lambda_i[4], 2) +      \
                    pow(lambda_i[5], 2));                                   \
        energy = c1F * ucl_sqrt(rho_i) + c2F * pow(rho_i, 2) +              \
                 0.5 * sum_mu + 0.5 * sum_lamb - 1.0 / 6.0 * v_i * v_i;     \
        if(EVFLAG && eflag)                                                 \
            engv[ii] = energy;                                              \
    }

#define local_allocate_store_answer_anna_adp()                              \
    __local acctyp red_acc[6][BLOCK_PAIR];

#define store_answer_anna_adp(f, energy, virial, ii, inum,                  \
                              tid, e_base, t_per_atom,                      \
                              offset, eflag, vflag, ans, engv)              \
    if (t_per_atom > 1) {                                                   \
        red_acc[0][tid] = f.x;                                              \
        red_acc[1][tid] = f.y;                                              \
        red_acc[2][tid] = f.z;                                              \
        red_acc[3][tid] = energy;                                           \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if (offset < s) {                                               \
                for (int n = 0; n < 4; n++)                                 \
                    red_acc[n][tid] += red_acc[n][tid + s];                 \
            }                                                               \
        }                                                                   \
        f.x = red_acc[0][tid];                                              \
        f.y = red_acc[1][tid];                                              \
        f.z = red_acc[2][tid];                                              \
        energy = red_acc[3][tid];                                           \
        if (EVFLAG && vflag) {                                              \
            simdsync();                                                     \
            for (int n = 0; n < 6; n++)                                     \
                red_acc[n][tid] = virial[n];                                \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    for (int n = 0; n < 6; n++)                             \
                        red_acc[n][tid] += red_acc[n][tid + s];             \
                }                                                           \
            }                                                               \
            for (int n = 0; n < 6; n++)                                     \
                virial[n] = red_acc[n][tid];                                \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        int ei = ii;                                                        \
        if (EVFLAG && eflag) {                                              \
            engv[ei] += 0.5 * energy + e_base;                              \
            ei += inum;                                                     \
        }                                                                   \
        if (EVFLAG && vflag) {                                              \
            for (int n = 0; n < 6; n++) {                                   \
                engv[ei] = virial[i] * (acctyp)0.5;                         \
                ei += inum;                                                 \
            }                                                               \
        }                                                                   \
        ans[ii] = f;                                                        \
    }

#else

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(dev_nbor, in_out, ii, nbor_pitch, num_in,                  \
                 num_ou, tid, t_per_atom, offset)                           \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 20] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            num_in += shfl_down(num_in, s, t_per_atom);                     \
        }                                                                   \
    } else {                                                                \
        in_out[offset] = num_in;                                            \
        in_out[offset + 20] = num_ou;                                       \
    }                                                                       \
    if(offset == 0) {                                                       \
        dev_nbor[ii + nbor_pitch] = num_in;                                 \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accGi[BLOCK_PAIR];

#define acc_Gi(Gi, nsf, tid, t_per_atom, offset)                            \
    if (t_per_atom > 1) {                                                   \
       for (int m = 0; m < nsf; m++) {                                      \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                G_i[m] += shfl_down(G_i[m], s, t_per_atom);                 \
            }                                                               \
            red_accGi[tid] = G_i[m];                                        \
            red_accGi[tid] = red_accGi[tid - offset];                       \
            G_i[m] = red_accGi[tid];                                        \
        }                                                                   \
    }

#define local_allocate_acc_hide()                                           \
    __local numtyp red_acchid[BLOCK_PAIR];

#define acc_hide(hid, t_hid, nnod, tid, t_per_atom, offset)                 \
    if(t_per_atom > 1) {                                                    \
        for(int i = 0; i < nnod; i++) {                                     \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                 t_hid[i] += shfl_down(t_hid[i], s, t_per_atom);            \
            }                                                               \
            red_acchid[tid] = t_hid[i];                                     \
            red_acchid[tid] = red_acchid[tid - offset];                     \
            hid[i] = red_acchid[tid];                                       \
        }                                                                   \
    }

#define local_allocate_store_energy_padp()

#define store_energy_padp(rho_i, mu_i, lambda_i, c1F, c2F, adp_rho,         \
                          adp_mu, adp_lambda, energy, ii, i, nall, inum,    \
                          tid, t_per_atom, offset, eflag, vflag, engv)      \
    if (t_per_atom > 1) {                                                   \
         for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {            \
            rho_i += shfl_down(rho_i, s, t_per_atom);                       \
        }                                                                   \
        for (int k = 0; k < 3; k++) {                                       \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                mu_i[k] += shfl_down(mu_i[k], s, t_per_atom);               \
           }                                                                \
        }                                                                   \
        for (int k = 0; k < 6; k++) {                                       \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                lambda_i[k] += shfl_down(lambda_i[k], s, t_per_atom);       \
           }                                                                \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        acctyp v_i = lambda_i[0] + lambda_i[1] + lambda_i[2];               \
        acctyp sum_mu = 0.0, sum_lamb = 0.0;                                \
        adp_rho[i] = rho_i;                                                 \
        for(int k = 0; k < 6; k++) {                                        \
            if(k < 3) {                                                     \
                sum_mu += mu_i[k] * mu_i[k];                                \
                sum_lamb += lambda_i[k] * lambda_i[k];                      \
                adp_mu[i + k * nall] = mu_i[k];                             \
            }                                                               \
            adp_lambda[i + k * nall] = lambda_i[k];                         \
        }                                                                   \
        sum_lamb += 2.0 * (pow(lambda_i[3], 2) + pow(lambda_i[4], 2) +      \
                    pow(lambda_i[5], 2));                                   \
        energy = c1F * ucl_sqrt(rho_i) + c2F * pow(rho_i, 2) +              \
                 0.5 * sum_mu + 0.5 * sum_lamb - 1.0 / 6.0 * v_i * v_i;     \
        if(EVFLAG && eflag)                                                 \
            engv[ii] =  energy;                                             \
    }

#define local_allocate_store_answer_anna_adp()

#define store_answer_anna_adp(f, energy, virial, ii, inum,                  \
                              tid, e_base, t_per_atom,                      \
                              offset, eflag, vflag, ans, engv)              \
    if(t_per_atom > 1) {                                                    \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            f.x += shfl_down(f.x, s, t_per_atom);                           \
            f.y += shfl_down(f.y, s, t_per_atom);                           \
            f.z += shfl_down(f.z, s, t_per_atom);                           \
            if (EVFLAG)                                                     \
                energy += shfl_down(energy, s, t_per_atom);                 \
        }                                                                   \
        if (EVFLAG && vflag) {                                              \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                for (int n = 0; n < 6; n++)                                 \
                    virial[n] += shfl_down(virial[n], s, t_per_atom);       \
            }                                                               \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        int ei = ii;                                                        \
        if (EVFLAG && eflag) {                                              \
            engv[ei] += 0.5 * energy + e_base;                              \
            ei += inum;                                                     \
        }                                                                   \
        if (EVFLAG && vflag) {                                              \
            for (int n = 0; n < 6; n++) {                                   \
                engv[ei] = virial[n] * (acctyp)0.5;                         \
                ei += inum;                                                 \
            }                                                               \
        }                                                                   \
        ans[ii] = f;                                                        \
    }

#endif

//---------------------------------------------------------------------
	                // get the short neighbor list
//----------------------------------------------------------------------
__kernel void k_anna_adp_short_nbor(const __global numtyp4* restrict x_, 
									const numtyp cutMax, const int ntypes, 
									__global int* dev_nbor, 
									const int nbor_pitch, 
									__global int* dev_packed, 
									const int inum, const int t_per_atom) {

	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);

	local_allocate_acc_numj();
	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;																
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
				  offset, i, jnum, n_stride, nbor_end, nbor_j);

		numtyp4 ix; fetch4(ix, i, pos_tex);
		int nbor_begin = nbor_j;																			
		int index_in = 0;																			
		int index_ou = 0;
		int j_out[300], j_in[300];																	
		
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int sj = dev_packed[nbor_j];
			int sj_nomask = sj;
			sj &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, sj, pos_tex);

			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;
			
			if (r2ij <= cutMax && r2ij > 1.0e-10) {
				j_in[index_in] = sj_nomask;
				index_in++;
			} else {
				j_out[index_ou] = sj_nomask;
				index_ou++;
			}
		}																								
		int in_out[40];																				
		acc_numj(dev_nbor, in_out, ii, nbor_pitch, index_in, 
				 index_ou, tid, t_per_atom, offset);												

		int sum_in = 0, sum_ou = dev_nbor[ii + nbor_pitch];
		nbor_begin -= offset;																		
		for (int j = 0; j < offset; j++) {
			sum_in += in_out[j];																	
			sum_ou += in_out[j + 20];
		}
		for (int j = 0; j < in_out[offset]; j++) {
			int index = sum_in + j;
			int begin_in = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_in] = j_in[j];
		}
		for (int j = 0; j < in_out[offset + 20]; j++) {
			int index = sum_ou + j;
			int begin_ou = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_ou] = j_out[j];
		}
	}
}

//---------------------------------------------------------------------
	 /* __kernel extern "C" __global__, in "ucl_nv_kernel.h" file
	    embed energy, and angular energy will be calculated here*/
//----------------------------------------------------------------------
__kernel void k_energy(const __global numtyp4* restrict x_, const __global int* restrict map,
					   const int inum, const int t_per_atom, const __global int* dev_nbor, 
					   const __global int* dev_packed, const int nbor_pitch, const int nall, 
					   const int ntl, const int nhl, const int nnod, const int npsf, 
					   const int ntsf, const int nsf, const int nout, const __global int* flagact, 
					   const int eflag, const int vflag, const numtyp2 adp_const,
					   const __global numtyp* restrict weight_all, 
					   const __global numtyp* restrict bias_all, 
					   const __global numtyp* gadp_params, __global numtyp* restrict adp_rho, 
					   __global numtyp* restrict adp_mu, __global numtyp* restrict adp_lambda, 
					   __global numtyp* restrict ladp_params, __global acctyp* restrict engv) {

	local_allocate_acc_Gi();
	local_allocate_acc_hide();
	local_allocate_store_energy_padp();																

	numtyp cutMax = adp_const.y;
	numtyp Rc = ucl_sqrt(cutMax);
	numtyp coeff_fc = MY_PI / Rc;																	
	acctyp energy = 0.0;
	numtyp A0 = gadp_params[0];
	numtyp yy = gadp_params[1];
	numtyp gamma = gadp_params[2];
	numtyp C0 = gadp_params[3];
	numtyp c1F = gadp_params[4];
	numtyp c2F = gadp_params[5];
	numtyp r0 = gadp_params[10];
	numtyp hc = gadp_params[12];
	numtyp d1 = gadp_params[13];
	numtyp q1 = gadp_params[14];
	numtyp d3 = gadp_params[15];
	numtyp q3 = gadp_params[16];	
	
	//------------------starting calculation-------------------
	int tid, ii, offset, n_stride;																	
	atom_info(t_per_atom, ii, tid, offset);															
	
	if (ii < inum) {
		int i, nbor_j0, nbor_j, nbor_end, jnum;
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, offset, i, jnum, n_stride, nbor_end, nbor_j0);			
		numtyp4 ix; fetch4(ix, i, pos_tex);
		numtyp G_i[28] = { 0.0 };

		//-----------------all neighbors-----------------
		int idj = offset;
		for (nbor_j = nbor_j0; nbor_j < nbor_end; nbor_j += n_stride, idj += t_per_atom) {
			int j = dev_packed[nbor_j];																
			j &= NEIGHMASK;

			numtyp4 jx, tx; fetch4(jx, j, pos_tex);													
			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;
			numtyp rij = ucl_sqrt(r2ij);

			//-----------------pair symmetry function-----------------
			numtyp xij = 2.0 * rij / Rc - 1.0;
			numtyp fcij = 0.5 * (cos(rij * coeff_fc) + 1.0);
			tx.x = (numtyp)1.0;
			tx.y = xij;
			G_i[0] += fcij;
			G_i[1] += fcij * tx.y;
			for (int m = 2; m < npsf; m++) {
				tx.z = 2.0 * xij * tx.y - tx.x;
				tx.x = tx.y;
				tx.y = tx.z;
				G_i[m] += fcij * tx.z;
			}

			//-----------------triple symmetry function-----------------
			int  nbor_k, nbor_kend, k_loop;
			k_loop = 1 + jnum / t_per_atom - (int)idj / t_per_atom;									
			nbor_k = nbor_j;
			nbor_kend = nbor_j + t_per_atom - offset;
			for (int n = 0; n < k_loop; n++) {
				if (n != 0) {
					nbor_k = nbor_j;
					nbor_k += (n * n_stride - offset);
					nbor_kend = nbor_k + t_per_atom;
				}
				if (nbor_kend > nbor_end)	nbor_kend = nbor_end;
				for (; nbor_k < nbor_kend; nbor_k++) {
					if (nbor_k == nbor_j)	continue;
					int k = dev_packed[nbor_k];
					k &= NEIGHMASK;
					numtyp4 kx; fetch4(kx, k, pos_tex);
					numtyp delt2x = ix.x - kx.x;
					numtyp delt2y = ix.y - kx.y;
					numtyp delt2z = ix.z - kx.z;
					numtyp r2ik = delt2x * delt2x + delt2y * delt2y + delt2z * delt2z;
					numtyp rik = ucl_sqrt(r2ik);
					numtyp rinv12 = ucl_recip(rij * rik);
					numtyp cos_theta = (deltx * delt2x + delty * delt2y + deltz * delt2z) * rinv12;
					
					numtyp fcik = 0.5 * (cos(rik * coeff_fc) + 1.0);
					numtyp xik = 0.5 * (cos_theta + 1.0);
					numtyp fcijk = fcij * fcik;
					tx.x = (numtyp)1.0;
					tx.y = xik;
					G_i[npsf] += fcijk;
					G_i[npsf + 1] += fcijk * tx.y;
					for (int m = 2; m < ntsf; m++) {
						tx.z = 2.0 * xik * tx.y - tx.x;
						tx.x = tx.y;
						tx.y = tx.z;
						G_i[npsf + m] += fcijk * tx.z;
					}	// G_triple
				}	// neigbor_k
			}	// k_loop
		}	// j_loop
		acc_Gi(G_i, nsf, tid, t_per_atom, offset);													

		//----------------- feedforward for local parameters -----------------
		int id_w = 0, n_row[3], n_col[3], w_begin[3], b_begin[3];
		acctyp weight, bias;
		acctyp hid[6] = { 0.0 };																	
		
		n_row[0] = nnod; n_col[0] = nsf;
		n_row[1] = nnod; n_col[1] = nnod;
		n_row[2] = nout; n_col[2] = nnod;
		b_begin[0] = 0;	 	
		b_begin[1] = n_row[0];
		b_begin[2] = b_begin[1] + n_row[1];
		w_begin[0] = 0;	
		w_begin[1] = n_row[0] * n_col[0];
		w_begin[2] = w_begin[1] + n_row[1] * n_col[1];

		for (int n = 0; n < ntl - 1; n++) {
			int row_loop = 1 + n_row[n] / t_per_atom;
			if(n_row[n] % t_per_atom == 0)
				row_loop -=1;
			int flag_act = flagact[n];
			numtyp t_hid[6] = { 0.0 };																

			for(int k = 0; k < row_loop; k++) {														
				int id = offset + k * t_per_atom;
				if(id < n_row[n]) {																	
					fetch(bias, id + b_begin[n], bias_tex);
					id_w = w_begin[n] + id * n_col[n];
					if(n == 0) {
						for(int m = 0; m < n_col[n]; m++) {
							//fetch(weight, id_w, weight_tex);										
							weight = weight_all[id_w];
							t_hid[id] += weight * G_i[m];
							id_w++;
						}
					} else {
						for(int m = 0; m < n_col[n]; m++) {
							//fetch(weight, id_w, weight_tex);										
							weight = weight_all[id_w];
							t_hid[id] += weight * hid[m];
							id_w++;
						}
					}
					t_hid[id] += bias;
					if(flag_act == 0)
						t_hid[id] = t_hid[id];
					if(flag_act == 4) {																
						numtyp exp_2x = ucl_exp(coeff_b * t_hid[id]);
						t_hid[id] = coeff_a * ((exp_2x - 1.0) / (exp_2x + 1.0));
					}
				}
			}
			acc_hide(hid, t_hid, n_row[n], tid, t_per_atom, offset);								
		}
		
		//----------------- energy calculation -----------------
		if(nout < t_per_atom && offset < nout) {
			ladp_params[i + offset * nall] = hid[offset];
		}
		acctyp d2 = hid[0];
		acctyp q2 = hid[1];																			

		acctyp rho_i = 0.0, mu_i[3] = { 0.0 }, lambda_i[6] = { 0.0 };								
		for (nbor_j = nbor_j0; nbor_j < nbor_end; nbor_j += n_stride) {
			int j = dev_packed[nbor_j];																
			j &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, j, pos_tex);														
			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;
			numtyp rij = ucl_sqrt(r2ij);

			numtyp stpf_x = (rij - Rc) / hc;
			numtyp stpf_x4 = stpf_x * stpf_x * stpf_x * stpf_x;
			numtyp adp_stpf = stpf_x4 / (1.0 + stpf_x4);
			numtyp adp_u = adp_stpf * (d1 * exp(-d2 * rij) + d3);
			numtyp adp_w = adp_stpf * (q1 * exp(-q2 * rij) + q3);

			mu_i[0] += adp_u * deltx;
			mu_i[1] += adp_u * delty;
			mu_i[2] += adp_u * deltz;

			lambda_i[0] += adp_w * deltx * deltx;
			lambda_i[1] += adp_w * delty * delty;
			lambda_i[2] += adp_w * deltz * deltz;
			lambda_i[3] += adp_w * deltx * delty;
			lambda_i[4] += adp_w * deltx * deltz;
			lambda_i[5] += adp_w * delty * deltz;

			numtyp rho_z = rij - r0;
			numtyp exp_z = exp(-gamma * rho_z);
			rho_i += adp_stpf * (A0 * pow(rho_z, yy) * exp_z * (1 + exp_z) + C0);
		}
		store_energy_padp(rho_i, mu_i, lambda_i, c1F, c2F, adp_rho, adp_mu, adp_lambda, 
						  energy, ii, i, nall, inum, tid, t_per_atom, offset, eflag, vflag, engv);
	}	// if ii
}

//----------------------------------------------------------------------
			// force of atom i and energy of pair part
//----------------------------------------------------------------------
__kernel void k_anna_adp(const __global numtyp4* restrict x_, const int ntypes, 
						 const int t_per_atom, const __global int* dev_nbor,
						 const __global int* dev_packed, const int nbor_pitch,
						 __global acctyp4 *ans, __global acctyp* restrict engv,
						 const int eflag, const int vflag, const int inum,
						 const __global numtyp* gadp_params, const numtyp2 adp_const,
						 const int nall, const __global numtyp* restrict adp_rho,
						 const __global numtyp* restrict adp_mu, 
						 const __global numtyp* restrict adp_lambda,
						 const __global numtyp* restrict ladp_params) {
	
	local_allocate_store_answer_anna_adp();
	numtyp e_base = adp_const.x;																
	numtyp cutMax = adp_const.y;
	numtyp Rc = ucl_sqrt(cutMax);
	numtyp A0 = gadp_params[0];
	numtyp yy = gadp_params[1];
	numtyp gamma = gadp_params[2];
	numtyp C0 = gadp_params[3];
	numtyp c1F = 0.5 * gadp_params[4];
	numtyp c2F = 2.0 * gadp_params[5];
	numtyp V0 = gadp_params[6];
	numtyp b1 = gadp_params[7];
	numtyp b2 = gadp_params[8];
	numtyp delta = gadp_params[9];
	numtyp r0 = gadp_params[10];
	numtyp r1 = gadp_params[11];
	numtyp hc = gadp_params[12];
	numtyp d1 = gadp_params[13];
	numtyp q1 = gadp_params[14];
	numtyp d3 = gadp_params[15];
	numtyp q3 = gadp_params[16];	

	acctyp4 f;	f.x = (acctyp)0.0; f.y = (acctyp)0.0; f.z = (acctyp)0.0;
	acctyp fx, fy, fz, fxi, fyi, fzi, fxj, fyj, fzj;
	acctyp energy = (acctyp)0.0;
	acctyp virial[6] = { 0.0 };																		

	//------------------ starting calculation -------------------
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);															
	numtyp rho_i, rho_j, mu_i[3], mu_j[3], lambda_i[6], lambda_j[6];
	numtyp d2_i, q2_i;
	numtyp d2_j, q2_j;

	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, offset, i, jnum, n_stride, nbor_end, nbor_j);		
		
		numtyp4 ix;		fetch4(ix, i, pos_tex);
		fetch(rho_i, i, rho_tex);	
		fetch(d2_i, i, ladp_tex);
		fetch(q2_i, i + nall, ladp_tex);

		for(int k = 0; k < 6; k++) {
			if(k < 3)
				fetch(mu_i[k], i + k * nall, imu_tex);
			fetch(lambda_i[k], i + k * nall, lambda_tex);
		}
		numtyp v_i = -1.0 / 3.0 * (lambda_i[0] + lambda_i[1] + lambda_i[2]);
		numtyp rep_coeff = V0 / (b2 - b1);
		numtyp x, y, z, r2ij, rij;

		// all neighbors
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int j = dev_packed[nbor_j];															
			j &= NEIGHMASK;

			fetch(rho_j, j, rho_tex);		
			fetch(d2_j, j, ladp_tex);
			fetch(q2_j, j + nall, ladp_tex);

			for(int k = 0; k < 6; k++) {
				if(k < 3)
					fetch(mu_j[k], j + k * nall, imu_tex);
				fetch(lambda_j[k], j + k * nall, lambda_tex);
			}
			numtyp v_j = -1.0 / 3.0 * (lambda_j[0] + lambda_j[1] + lambda_j[2]);
			
			numtyp4 jx; fetch4(jx, j, pos_tex);														
			x = ix.x - jx.x;
			y = ix.y - jx.y;
			z = ix.z - jx.z;
			r2ij = x * x + y * y + z * z;
			rij = ucl_sqrt(r2ij);

			numtyp stpf_x = (rij - Rc) / hc;
			numtyp stpf_x3 = stpf_x * stpf_x * stpf_x;
			numtyp stpf_t1 = 1.0 + stpf_x * stpf_x3;
			numtyp stpf = stpf_x * stpf_x3 / stpf_t1;
			numtyp dstpf = 4.0 * stpf_x3 / stpf_t1 / stpf_t1 /hc;

            numtyp rho_z = rij - r0;
			numtyp exp_z = exp(-gamma * rho_z);
			numtyp z_yy = A0 * pow(rho_z, yy);
			numtyp ga_zyy = z_yy * gamma;
			numtyp drho = exp_z * (1.0 + exp_z) * (z_yy * (dstpf + stpf * yy / rho_z) - ga_zyy) + C0 * dstpf - ga_zyy * exp_z * exp_z;
			
			numtyp dfp_i = (c1F * pow(rho_i, -0.5) + c2F * rho_i) * drho;
			numtyp dfp_j = (c1F * pow(rho_j, -0.5) + c2F * rho_j) * drho;
				
			numtyp repul_z = rij / r1;
			numtyp zb1 = pow(repul_z, b1);
			numtyp zb2 = pow(repul_z, b2);
			numtyp drep_t = b2 * b1 / r1;
			numtyp rep_t1 = rep_coeff * (b2 / zb1 - b1 / zb2) + delta;
			numtyp drep = dstpf * rep_t1 + stpf * rep_coeff * (drep_t / repul_z * (-1.0 / zb1 + 1.0 / zb2));

			numtyp u_t_i = d1 * exp(-d2_i * rij);
			numtyp w_t_i = q1 * exp(-q2_i * rij);
			numtyp u_t_j = d1 * exp(-d2_j * rij);
			numtyp w_t_j = q1 * exp(-q2_j * rij);
			numtyp u_i = stpf * (u_t_i + d3);
			numtyp w_i = 2.0 * stpf * (w_t_i + q3);
			numtyp u_j = stpf * (u_t_j + d3);
			numtyp w_j = 2.0 * stpf * (w_t_j + q3);

			numtyp du_i = dstpf * (u_t_i + d3) - stpf * d2_i * u_t_i;
			numtyp dw_i = dstpf * (w_t_i + q3) - stpf * q2_i * w_t_i;
			numtyp du_j = dstpf * (u_t_j + d3) - stpf * d2_j * u_t_j;
			numtyp dw_j = dstpf * (w_t_j + q3) - stpf * q2_j * w_t_j;

			numtyp x2 = x * x, y2 = y * y, z2 = z * z, xy = x * y, xz = x * z, yz = y * z;
			numtyp dang_lamb1_i = dw_i * (lambda_i[0] * x2 + lambda_i[1] * y2 + lambda_i[2] * z2);
			numtyp dang_lamb2_i = dw_i * (lambda_i[3] * xy + lambda_i[4] * xz + lambda_i[5] * yz) * 2.0 + dang_lamb1_i;
			numtyp dang_lamb1_j = dw_j * (lambda_j[0] * x2 + lambda_j[1] * y2 + lambda_j[2] * z2);
			numtyp dang_lamb2_j = dw_j * (lambda_j[3] * xy + lambda_j[4] * xz + lambda_j[5] * yz) * 2.0 + dang_lamb1_j;
			numtyp df_t1_i = 0.5 * drep + dfp_i + du_i * (mu_i[0] * x + mu_i[1] * y + mu_i[2] * z) + dang_lamb2_i;
			numtyp df_t1_j = 0.5 * drep + dfp_j - du_j * (mu_j[0] * x + mu_j[1] * y + mu_j[2] * z) + dang_lamb2_j;
            numtyp df_t2_i = v_i * (dw_i * rij + w_i);
            numtyp df_t2_j = v_j * (dw_j * rij + w_j);

			// force for: atom i is the central atom
            fxi = df_t1_i * x / rij + w_i * (y * lambda_i[3] + z * lambda_i[4] + x * lambda_i[0]) + mu_i[0] * u_i + x * df_t2_i;
            fyi = df_t1_i * y / rij + w_i * (y * lambda_i[1] + z * lambda_i[5] + x * lambda_i[3]) + mu_i[1] * u_i + y * df_t2_i;
            fzi = df_t1_i * z / rij + w_i * (y * lambda_i[5] + z * lambda_i[2] + x * lambda_i[4]) + mu_i[2] * u_i + z * df_t2_i;
	
			// force for: atom j is the central atom
			fxj = -df_t1_j * x / rij - w_j * (y * lambda_j[3] + z * lambda_j[4] + x * lambda_j[0]) + mu_i[0] * u_j - x * df_t2_j;
            fyj = -df_t1_j * y / rij - w_j * (y * lambda_j[1] + z * lambda_j[5] + x * lambda_j[3]) + mu_i[1] * u_j - y * df_t2_j;
            fzj = -df_t1_j * z / rij - w_j * (y * lambda_j[5] + z * lambda_j[2] + x * lambda_j[4]) + mu_i[2] * u_j - z * df_t2_j;
			fx = fxj - fxi;
			fy = fyj - fyi;
			fz = fzj - fzi;

			f.x += fx;
			f.y += fy;
			f.z += fz;
			if (EVFLAG && eflag) {
				energy += stpf * rep_t1;
			}
			if (EVFLAG && vflag) {
				virial[0] += x * fx;
				virial[1] += y * fy;
				virial[2] += z * fz;
				virial[3] += x * fy;
				virial[4] += x * fz;
				virial[5] += y * fz;
			}
		}
		store_answer_anna_adp(f, energy, virial, ii, inum, tid, e_base, t_per_atom, offset, eflag, vflag, ans, engv);
	}
}
