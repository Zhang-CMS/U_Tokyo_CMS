#include "hip/hip_runtime.h"
//* Device code *---------------------------------------
//      Artifical Neural Network Potential
//             Accelerated by GPU
//______________________________________________________        
//  begin:  Wed February 16, 2022
//  email:  meng_zhang@metall.t.u-tokyo.ac.jp
//          junya_inoue@metall.t.u-tokyo.ac.jp 
//______________________________________________________
//------------------------------------------------------

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( weight_tex,float);
_texture( bias_tex,float);
_texture( sfsc_tex,float);
_texture( sfav_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( weight_tex,int2);
_texture( bias_tex,int2);
_texture( sfsc_tex,int2);
_texture( sfav_tex,int2);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sfnor_scal
#define sfav_tex sfnor_avg
#endif

#else
#define pos_tex x_
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sfnor_scal
#define sfav_tex sfnor_avg
#endif

#define MY_PI (numtyp)3.14159265358979323846
#define coeff_a (numtyp)1.7159
#define coeff_b (numtyp)0.666666666666667
#define coeff_c (numtyp)0.1

#if (SHUFFLE_AVAIL == 0)

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 10] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if (offset < s) {                                               \
                red_accj_in[tid] += red_accj_in[tid + s];                   \
            }                                                               \
        }                                                                   \
        num_in = red_accj_in[tid];                                          \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 10] = num_ou;                                       \
    }                                                                       \
    if(offset ==0) {                                                        \
        newj[ii] = num_in;                                                  \
    }


#define local_allocate_acc_dGij()                                           \
    __local numtyp red_accj[19][BLOCK_PAIR];

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k,                     \
                 ntsf, tid, offset, t_per_atom)                             \
    for (int m = 0; m < ntsf; m++) {                                        \
        red_acc[m][tid] = dG_dk[m].z;                                       \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                dG_dky[m][tid] += dG_dky[m][tid + s];                       \
                red_acc[m][tid] += red_acc[m][tid + s];                     \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            dGij[index_bm].y += dG_dky[m][tid];                             \
            dGij[index_bm].z += red_acc[m][tid];                            \
        }                                                                   \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    red_accG[tid] += red_accG[tid + s];                     \
                }                                                           \
            }                                                               \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }                                                                       \
    numtyp sfsc, sfav;                                                      \
    for (int i = 0; i < nsf; i++) {                                         \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfav, i, sfav_tex);                                           \
        dG_dj[i].w = dG_dj[i].w - sfsc * sfav;                              \
    }                                                                       \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            red_accG[tid] = red_accG[tid - offset];                         \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }

#define local_allocate_store_answers_annp()                                 \
    __local acctyp red_acc[3][BLOCK_PAIR];

#define store_answers_annp(fi, energy, ii, inum, tid, t_per_atom,           \
                           offset, eflag, engv, force)                      \
    if (t_per_atom > 1) {                                                   \
        red_acc[0][tid] = fi.x;                                             \
        red_acc[1][tid] = fi.y;                                             \
        red_acc[2][tid] = fi.z;                                             \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    for (int i = 0; i < 4; i++)                             \
                        red_acc[i][tid] += red_acc[i][tid + s];             \
                }                                                           \
        }                                                                   \
        fi.x = red_acc[0][tid];                                             \
        fi.y = red_acc[1][tid];                                             \
        fi.z = red_acc[2][tid];                                             \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        int ei = ii;                                                        \
        if (EVFLAG && eflag) {                                              \
            engv[ei] = energy;                                              \
        }                                                                   \
        acctyp4 old_f = force[ii];                                          \
        old_f.x += fi.x;                                                    \
        old_f.y += fi.y;                                                    \
        old_f.z += fi.z;                                                    \
        force[ii] = old_f;                                                  \
    }

#else

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 10] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            num_in += shfl_down(num_in, s, t_per_atom);                     \
        }                                                                   \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 10] = num_ou;                                       \
    }                                                                       \
    if (offset == 0) {                                                      \
        newj[ii] = num_in;                                                  \
    }

#define local_allocate_acc_dGij()

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k,                     \
                 ntsf, tid, offset, t_per_atom)                             \
    for (int m = 0; m < ntsf; m++) {                                        \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            dG_dkz[m] += shfl_down(dG_dkz[m], s, t_per_atom);               \
            simdsync();                                                     \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                dG_dky[m][tid] += dG_dky[m][tid + s];                       \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            dGij[index_bm].y += dG_dky[m][tid];                             \
            dGij[index_bm].z += dG_dkz[m];                                  \
        }                                                                   \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    if (t_per_atom > 1) {                                                   \
       for (int m = 0; m < nsf; m++) {	                                    \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                dG_dj[m].w += shfl_down(dG_dj[m].w, s, t_per_atom);         \
                simdsync();                                                 \
            }                                                               \
        }                                                                   \
    }                                                                       \
    numtyp sfsc, sfav;                                                      \
    for(int i = 0; i < nsf; i++) {                                          \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfav, i, sfav_tex);                                           \
        dG_dj[i].w = dG_dj[i].w - sfsc * sfav;                              \
    }                                                                       \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            red_accG[tid] = red_accG[tid - offset];                         \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }

#define local_allocate_store_answers_annp()

#define store_answers_annp(fi, energy, ii, inum, tid, t_per_atom,           \
                           offset, eflag, engv, force)                      \
    if (t_per_atom > 1) {                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            fi.x += shfl_down(fi.x, s, t_per_atom);                         \
            fi.y += shfl_down(fi.y, s, t_per_atom);                         \
            fi.z += shfl_down(fi.z, s, t_per_atom);                         \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii < inum) {                                         \
        int ei = ii;                                                        \
        if (EVFLAG && eflag) {                                              \
            engv[ei] = energy;                                              \
        }                                                                   \
        acctyp4 old_f = force[ii];                                          \
        old_f.x += fi.x;                                                    \
        old_f.y += fi.y;                                                    \
        old_f.z += fi.z;                                                    \
        force[ii] = old_f;                                                  \
    }

#endif

//---------------------------------------------------------------------
	// get the short neighbor list
//----------------------------------------------------------------------
__kernel void k_annp_short_nbor(const __global numtyp4* restrict x_,
								const __global numtyp* restrict cutsq,
								const int ntypes, 
								__global int* dev_nbor,
								__global int* dev_packed,
								__global int* newj, const int inum, 
								const int nbor_pitch, const int t_per_atom) {
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);
	local_allocate_acc_numj();

	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;																
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
				  offset, i, jnum, n_stride, nbor_end, nbor_j);

		numtyp4 ix; fetch4(ix, i, pos_tex);
		int nbor_begin = nbor_j;
		int itype = ix.w;																			
		int index_in = 0;																			
		int index_ou = 0;
		int j_out[300], j_in[300];																	
		
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int sj = dev_packed[nbor_j];
			int sj_nomask = sj;
			sj &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, sj, pos_tex);
			int jtype = jx.w;

			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp rsqij = deltx * deltx + delty * delty + deltz * deltz;

			int ijtype = itype * ntypes + jtype;
			if (rsqij <= cutsq[ijtype]) {
				j_in[index_in] = sj_nomask;
				index_in++;
			}
			else {
				j_out[index_ou] = sj_nomask;
				index_ou++;
			}
		}																										
		int in_out[20];																				
		acc_numj(newj, in_out, ii, index_in, index_ou, tid, t_per_atom, offset);					
		int numj = newj[ii];
		dev_nbor[ii + nbor_pitch] = numj;															

		int sum_in = 0, sum_ou = numj;
		nbor_begin -= offset;																		
		for (int j = 0; j < offset; j++) {
			sum_in += in_out[j];
			sum_ou += in_out[j + 10];
		}
		for (int j = 0; j < in_out[offset]; j++) {
			int index = sum_in + j;
			int begin_in = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_in] = j_in[j];
		}
		for (int j = 0; j < in_out[offset + 10]; j++) {
			int index = sum_ou + j;
			int begin_ou = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_ou] = j_out[j];
		}

		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii,
				  offset, i, jnum, n_stride, nbor_end, nbor_j);
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int sj = dev_packed[nbor_j];
			int sj_nomask = sj;
			sj &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, sj, pos_tex);
		}
	}
}

//---------------------------------------------------------------------
	// __kernel extern "C" __global__, in "ucl_nv_kernel.h" file
//----------------------------------------------------------------------
__kernel void k_annp(const __global numtyp4* restrict x_, const int ntypes,
				 	 const int ntl, const int nhl, const int nnod,
					 const int nsf, const int npsf, const int ntsf,
					 const __global int* restrict map, const int t_per_atom,
					 const __global numtyp* restrict cutsq,
					 const __global numtyp* restrict sfnor_scal,
					 const __global numtyp* restrict sfnor_avg,
					 const numtyp4 out_mod, const int eflag,
					 const __global numtyp* restrict weight_all,
					 const __global numtyp* restrict bias_all, const int inum,
					 const __global int* flagact, const __global int* dev_nbor,
					 const __global int* dev_packed, const int nbor_pitch,
					 __global acctyp4* Fj, 
					 __global acctyp* restrict engv, 
					 __global acctyp4* restrict force, 
					 __global numtyp4* dGij,
					 const int2 gpup, const int begin_i, const __global int* restrict newj) {

	int max_nbor_size = gpup.x;
	numtyp e_scale = out_mod.x;																		
	numtyp e_shift = out_mod.y;
	numtyp e_atom = out_mod.z;

	local_allocate_acc_dGij();
	local_allocate_acc_Gi();
	local_allocate_store_answers_annp();
	acctyp4 f;
	f.x = (acctyp)0; f.y = (acctyp)0; f.z = (acctyp).0;
	acctyp energy;
	energy = (acctyp)0;

	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);																

	int begin_jk = ii * max_nbor_size;																
	int index_bm;
	ii += begin_i;																					
	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum, n_jnum;
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
			      offset, i, jnum, n_stride, nbor_end, nbor_j);										

		numtyp4 ix; fetch4(ix, i, pos_tex);															
		int itype = ix.w;
		int idj = offset;																			
		numtyp4 dG_dj[28];

		numtyp dG_dkz[19];																			
		__shared__ numtyp dG_dkx[19][BLOCK_PAIR];
		__shared__ numtyp dG_dky[19][BLOCK_PAIR];
		for (int k = 0; k < 28; k++) {
			dG_dj[k].w = (numtyp)0.0;																
		}

		for (; nbor_j < nbor_end; nbor_j += n_stride, idj += t_per_atom) {
			for (int k = 0; k < 28; k++) {															
				dG_dj[k].x = (numtyp)0.0;
				dG_dj[k].y = (numtyp)0.0;
				dG_dj[k].z = (numtyp)0.0;
			}
			int j = dev_packed[nbor_j];																
			j &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, j, pos_tex);														
			int jtype = jx.w;
			int ijtype = itype * ntypes + jtype;
			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp rsqij = deltx * deltx + delty * delty + deltz * deltz;

			numtyp Rc_ij = ucl_sqrt(cutsq[ijtype]);
			numtyp rij = ucl_sqrt(rsqij);
			numtyp xij = 2.0 * rij / Rc_ij - 1.0;
			numtyp term1 = MY_PI * rij / Rc_ij;
			numtyp fcij = 0.5 * (cos(term1) + 1.0);
			numtyp dfcij = -0.5 * MY_PI * sin(term1) / Rc_ij;
			numtyp4 tx;
			numtyp4 dtx;
			numtyp sf_scal;
			tx.x = (numtyp)1.0;						tx.y = xij;
			dtx.x = (numtyp)0.0;					dtx.y = (numtyp)1.0;
			numtyp4 term_fc;
			term_fc.x = 2.0 * fcij / Rc_ij;

			fetch(sf_scal, 0, sfsc_tex);															
			dG_dj[0].w += sf_scal * fcij;
			term1 = -dfcij * sf_scal / rij;
			dG_dj[0].x += term1 * deltx;			dG_dj[0].y += term1 * delty;
			dG_dj[0].z += term1 * deltz;

			fetch(sf_scal, 1, sfsc_tex);															
			dG_dj[1].w += sf_scal * fcij * xij;
			term1 = -sf_scal * (term_fc.x + xij * dfcij) / rij;
			dG_dj[1].x += term1 * deltx;			dG_dj[1].y += term1 * delty;
			dG_dj[1].z += term1 * deltz;

			for (int m = 2; m < npsf; m++) {
				fetch(sf_scal, m, sfsc_tex);
				tx.z = 2.0 * xij * tx.y - tx.x;
				dtx.z = 2.0 * tx.y + 2.0 * xij * dtx.y - dtx.x;
				tx.x = tx.y;
				dtx.x = dtx.y;
				tx.y = tx.z;
				dtx.y = dtx.z;

				dG_dj[m].w += sf_scal * fcij * tx.z;												
				term1 = -sf_scal * (dtx.z * term_fc.x + tx.z * dfcij) / rij;
				dG_dj[m].x += term1 * deltx;		dG_dj[m].y += term1 * delty;
				dG_dj[m].z += term1 * deltz;
			}

			int idk = idj, nbor_k, nbor_kend, k_loop;
			k_loop = 1 + jnum / t_per_atom - (int)idj / t_per_atom;
			nbor_k = nbor_j;
			nbor_kend = nbor_j + t_per_atom - offset;
			for (int n = 0; n < k_loop; n++) {
				if (n != 0) {
					nbor_k = nbor_j;
					nbor_k += (n * n_stride - offset);
					nbor_kend = nbor_k + t_per_atom;
				}
				if (nbor_kend > nbor_end)	nbor_kend = nbor_end;
				for (; nbor_k < nbor_kend; nbor_k++, idk++) {
					if (nbor_k == nbor_j)	continue;
					for (int m = 0; m < 19; m++) {													
						dG_dkx[m][tid] = (numtyp)0.0;
						dG_dky[m][tid] = (numtyp)0.0;
						dG_dkz[m] = (numtyp)0.0;
					}

					int k = dev_packed[nbor_k];
					k &= NEIGHMASK;
					numtyp4 kx; fetch4(kx, k, pos_tex);
					int ktype = kx.w;
					int iktype = itype * ntypes + ktype;
					numtyp delt2x = ix.x - kx.x;
					numtyp delt2y = ix.y - kx.y;
					numtyp delt2z = ix.z - kx.z;
					numtyp rsqik = delt2x * delt2x + delt2y * delt2y + delt2z * delt2z;

					numtyp Rc_ik = ucl_sqrt(cutsq[iktype]);
					numtyp rik = ucl_sqrt(rsqik);
					numtyp rinv12 = ucl_recip(rij * rik);
					numtyp cos_theta = (deltx * delt2x + delty * delt2y + deltz * delt2z) * rinv12;
					numtyp xik = 0.5 * (cos_theta + 1.0);
					term1 = MY_PI * rik / Rc_ik;
					numtyp fcik = 0.5 * (cos(term1) + 1.0);
					numtyp dfcik = -0.5 * MY_PI * sin(term1) / Rc_ik;

					numtyp4 dct_dj;
					numtyp4 dct_dk;
					numtyp4 tdGt_dj;
					numtyp4 tdGt_dk;
					term1 = cos_theta / rsqij;
					numtyp term2 = cos_theta / rsqik;
					dct_dj.x = -delt2x * rinv12 + term1 * deltx;
					dct_dj.y = -delt2y * rinv12 + term1 * delty;
					dct_dj.z = -delt2z * rinv12 + term1 * deltz;
					dct_dk.x = -deltx * rinv12 + term2 * delt2x;
					dct_dk.y = -delty * rinv12 + term2 * delt2y;
					dct_dk.z = -deltz * rinv12 + term2 * delt2z;
					tx.x = (numtyp)1.0;							tx.y = xik;
					dtx.x = (numtyp)0.0;						dtx.y = (numtyp)1.0;
					term_fc.x = fcij * fcik;
					term_fc.y = dfcij * fcik / rij;
					term_fc.z = dfcik * fcij / rik;

					fetch(sf_scal, npsf, sfsc_tex);
					dG_dj[npsf].w += sf_scal * term_fc.x;
					term2 = sf_scal * term_fc.y;													
					numtyp term3 = sf_scal * term_fc.z;
					tdGt_dj.x = -term2 * deltx;
					tdGt_dj.y = -term2 * delty;
					tdGt_dj.z = -term2 * deltz;
					tdGt_dk.x = -term3 * delt2x;
					tdGt_dk.y = -term3 * delt2y;
					tdGt_dk.z = -term3 * delt2z;
					dG_dj[npsf].x += tdGt_dj.x;					dG_dj[npsf].y += tdGt_dj.y;
					dG_dj[npsf].z += tdGt_dj.z;
					dG_dkx[0][tid] += tdGt_dk.x;				dG_dky[0][tid] += tdGt_dk.y;
					dG_dkz[0] += tdGt_dk.z;	

					int index_t = npsf + 1;
					fetch(sf_scal, index_t, sfsc_tex);
					dG_dj[index_t].w += sf_scal * tx.y * term_fc.x;
					term1 = 0.5 * sf_scal * term_fc.x;
					term2 = sf_scal * tx.y * term_fc.y;
					term3 = sf_scal * tx.y * term_fc.z;
					tdGt_dj.x = term1 * dct_dj.x - term2 * deltx;
					tdGt_dj.y = term1 * dct_dj.y - term2 * delty;
					tdGt_dj.z = term1 * dct_dj.z - term2 * deltz;
					tdGt_dk.x = term1 * dct_dk.x - term3 * delt2x;
					tdGt_dk.y = term1 * dct_dk.y - term3 * delt2y;
					tdGt_dk.z = term1 * dct_dk.z - term3 * delt2z;

					dG_dj[index_t].x += tdGt_dj.x;				dG_dj[index_t].y += tdGt_dj.y;
					dG_dj[index_t].z += tdGt_dj.z;
					dG_dkx[1][tid] += tdGt_dk.x;				dG_dky[1][tid] += tdGt_dk.y;
					dG_dkz[1] += tdGt_dk.z;

					for (int m = 2; m < ntsf; m++) {
						index_t = m + npsf;
						tx.z = 2.0 * xik * tx.y - tx.x;
						dtx.z = 2.0 * tx.y + 2.0 * xik * dtx.y - dtx.x;
						tx.x = tx.y;
						dtx.x = dtx.y;
						tx.y = tx.z;
						dtx.y = dtx.z;

						fetch(sf_scal, index_t, sfsc_tex);
						dG_dj[index_t].w += sf_scal * tx.z * term_fc.x;
						term1 = 0.5 * sf_scal * dtx.z * term_fc.x;
						term2 = sf_scal * tx.z * term_fc.y;
						term3 = sf_scal * tx.z * term_fc.z;
						tdGt_dj.x = term1 * dct_dj.x - term2 * deltx;
						tdGt_dj.y = term1 * dct_dj.y - term2 * delty;
						tdGt_dj.z = term1 * dct_dj.z - term2 * deltz;
						tdGt_dk.x = term1 * dct_dk.x - term3 * delt2x;
						tdGt_dk.y = term1 * dct_dk.y - term3 * delt2y;
						tdGt_dk.z = term1 * dct_dk.z - term3 * delt2z;

						dG_dj[index_t].x += tdGt_dj.x;			dG_dj[index_t].y += tdGt_dj.y;
						dG_dj[index_t].z += tdGt_dj.z;
						dG_dkx[m][tid] += tdGt_dk.x;			dG_dky[m][tid] += tdGt_dk.y;
						dG_dkz[m] += tdGt_dk.z;
					}																				

					int begin_k = (begin_jk + idk) * nsf + npsf;
					if (n == 0 || t_per_atom == 1)													
						for (int m = 0; m < ntsf; m++) {
							index_bm = begin_k + m;
							dGij[index_bm].x += dG_dkx[m][tid];
							dGij[index_bm].y += dG_dky[m][tid];
							dGij[index_bm].z += dG_dkz[m];
						}
					else {
						acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k, ntsf, tid, offset, t_per_atom);				
					}
				}
			}
			int begin_j = (begin_jk + idj) * nsf;
			dGij[begin_j].w = (numtyp)j;
			for (int m = 0; m < nsf; m++) {
				index_bm = begin_j + m;
				dGij[index_bm].x += dG_dj[m].x;
				dGij[index_bm].y += dG_dj[m].y;
				dGij[index_bm].z += dG_dj[m].z;
			}
		}																							
		acc_Gi(dG_dj, sfnor_scal, sfnor_avg, nsf, tid, t_per_atom, offset);

		numtyp hidly[10];																			
		numtyp t_hidly[10];
		numtyp hidly_d[10];																			
		numtyp lays_dw[10 * 28];																	
		numtyp temp_dw[10 * 28];
		numtyp hidly_dw[10 * 28];																												
		numtyp weight, bias;
		int index_w, index_w2;
		for (int m = 0; m < nnod; m++) {
			hidly[m] = 0.0;
			hidly_d[m] = 0.0;
			index_w = m * nsf;
			for (int n = 0; n < nsf; n++) {
				index_w2 = index_w + n;
				hidly_dw[index_w2] = 0.0;															
				temp_dw[index_w2] = 0.0;
			}
		}
		index_w = 0;
		index_w2 = 0;
		int2 nrc_w[3];
		int index_t;
		nrc_w[0].x = nnod; nrc_w[0].y = nsf;
		nrc_w[1].x = nnod; nrc_w[1].y = nnod;
		nrc_w[2].x = 1; nrc_w[2].y = nnod;
		for (int n = 0; n < ntl - 1; n++) {
			for (int k = 0; k < nnod; k++) {
				t_hidly[k] = 0.0;
				index_t = k * nsf;
				for (int m = 0; m < nsf; m++) {														
					lays_dw[index_t + m] = 0.0;
				}
			}
			int actflag = flagact[n];
			for (int k = 0; k < nrc_w[n].x; k++) {
				fetch(bias, k + n * nnod, bias_tex);
				for (int m = 0; m < nrc_w[n].y; m++) {
					weight = weight_all[index_w];
					if (n == 0) {
						t_hidly[k] += weight * dG_dj[m].w;
					}
					else {
						t_hidly[k] += weight * hidly[m];
					}
					index_w++;
				}
				t_hidly[k] += bias;
			}
			for (int k = 0; k < nrc_w[n].x; k++) {													 
				if (actflag == 0) {
					t_hidly[k] = t_hidly[k];
					hidly_d[k] = 1;
				}
				if (actflag == 4) {
					numtyp t_exp = coeff_b * t_hidly[k];
					numtyp t_tanhx = (ucl_exp(t_exp) - ucl_exp(-t_exp)) / (ucl_exp(t_exp) + ucl_exp(-t_exp));
					t_hidly[k] = coeff_a * t_tanhx + coeff_c * t_hidly[k];
					hidly_d[k] = coeff_a * (1.0 - t_tanhx * t_tanhx) * coeff_b + coeff_c;
				}
				hidly[k] = t_hidly[k];
			}
			int index_dw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {													
				for (int m = 0; m < nrc_w[n].y; m++) {										
					weight = weight_all[index_w2];
					hidly_dw[index_dw] = hidly_d[k] * weight;
					index_w2++;
					index_dw++;
				}
			}																						
			int index_tdw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {
				for (int m = 0; m < nsf; m++) {
					if (n == 0) {
						temp_dw[k * nsf + m] = hidly_dw[k * nsf + m];
					}
					else {
						for (int j = 0; j < nrc_w[n].y; j++) {
							lays_dw[index_tdw] += hidly_dw[k * nnod + j] * temp_dw[j * nsf + m];
						}
						index_tdw++;
					}
				}
			}
			if (n != 0 && n != ntl - 1)																
				for (int k = 0; k < nrc_w[n].x; k++) {
					for (int m = 0; m < nsf; m++) {
						temp_dw[k * nsf + m] = lays_dw[k * nsf + m];
					}
				}																					
		}
		if (offset == 0) {
			engv[ii] = e_scale * hidly[0] + e_shift + e_atom;
		}

		acctyp4 F;
		for (int jj = offset; jj < jnum; jj += t_per_atom) {
			F.x = 0.0; F.y = 0.0; F.z = 0.0;
			int begin_j = (begin_jk + jj) * nsf;
			for (int k = 0; k < nsf; k++) {
				F.x -= lays_dw[k] * dGij[begin_j + k].x * e_scale;
				F.y -= lays_dw[k] * dGij[begin_j + k].y * e_scale;
				F.z -= lays_dw[k] * dGij[begin_j + k].z * e_scale;
			}
			Fj[begin_jk + jj].x += F.x;
			Fj[begin_jk + jj].y += F.y;
			Fj[begin_jk + jj].z += F.z;
			Fj[begin_jk + jj].w = dGij[begin_j].w;
		}
	}																								
}

//----------------------------------------------------------------------
	// updating the force for neighbor
//----------------------------------------------------------------------
__kernel void k_annp_updat(const __global int* restrict newj, 
						   const __global acctyp4* restrict Fj,
						   __global acctyp4* force,
						   const int begin_i, const int2 gpup) {

	int max_nbor_size = gpup.x;
	int num_atoms = gpup.y;
	int tid = THREAD_ID_X;
	__shared__ int ii;

	for (ii = 0; ii < num_atoms; ) {
		__shared__ acctyp4 tFj[BLOCK_PAIR];
		int n_jnum = newj[ii + begin_i];
		int begin_jk = ii * max_nbor_size;
		if (tid < n_jnum) {
			int idj = begin_jk + tid;
			int index = (int)Fj[idj].w;
			tFj[tid].x = Fj[idj].x;
			tFj[tid].y = Fj[idj].y;
			tFj[tid].z = Fj[idj].z;

			acctyp4 old_f = force[index];															
			old_f.w = index;
			old_f.x += tFj[tid].x;
			old_f.y += tFj[tid].y;
			old_f.z += tFj[tid].z;
			force[index] = old_f;
		}
		for (unsigned int s = n_jnum / 2; s > 0; s >>= 1) {
			__syncthreads();
			if (tid < s) {
				tFj[tid].x += tFj[tid + s].x;
				tFj[tid].y += tFj[tid + s].y;
				tFj[tid].z += tFj[tid + s].z;
			}
			__syncthreads();
			if (s % 2 == 1 && s != 1 && tid == 0) {													
				tFj[tid].x += tFj[tid + s - 1].x;
				tFj[tid].y += tFj[tid + s - 1].y;
				tFj[tid].z += tFj[tid + s - 1].z;
			}
		}

		if (n_jnum % 2 == 1 && tid == 0) {															
			tFj[tid].x += tFj[tid + n_jnum - 1].x;
			tFj[tid].y += tFj[tid + n_jnum - 1].y;
			tFj[tid].z += tFj[tid + n_jnum - 1].z;
		}
		if (tid == 0) {
			acctyp4 old_f = force[ii + begin_i];													
			old_f.w = (numtyp)(ii + begin_i);
			old_f.x -= tFj[tid].x;
			old_f.y -= tFj[tid].y;
			old_f.z -= tFj[tid].z;
			force[ii + begin_i] = old_f;
		}
		__syncthreads();		
		ii++;
	}
}
