#include "hip/hip_runtime.h"
//* Device code *---------------------------------------
//      Artifical Neural Network Potential
//             Accelerated by GPU
//______________________________________________________        
//  begin:  Mon Oct 23, 2022
//  email:  meng_zhang@metall.t.u-tokyo.ac.jp
//          junya_inoue@metall.t.u-tokyo.ac.jp 
//______________________________________________________
//------------------------------------------------------

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( weight_tex,float);
_texture( bias_tex,float);
_texture( sfsc_tex,float);
_texture( sfmi_tex,float);
_texture( cofsym_tex, float4);
#else
_texture_2d( pos_tex,int4);
_texture( weight_tex,int2);
_texture( bias_tex,int2);
_texture( sfsc_tex,int2);
_texture( sfmi_tex,int2);
_texture( cofsym_tex, int4);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sf_scal
#define sfmi_tex sf_min
#define cofsym_tex coeff_sym
#endif

#else
#define pos_tex x_
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sf_scal
#define sfmi_tex sf_min
#define cofsym_tex coeff_sym
#endif

#define MY_PI (numtyp)3.14159265358979323846
#define coeff_a (numtyp)1.7159
#define coeff_b (numtyp)0.666666666666667
#define coeff_c (numtyp)0.1
#define CFLENGTH (numtyp)1.889726
#define CFFORCE (numtyp)51.422515

#if (SHUFFLE_AVAIL == 0)

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 35] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if (offset < s) {                                               \
                red_accj_in[tid] += red_accj_in[tid + s];                   \
            }                                                               \
        }                                                                   \
        num_in = red_accj_in[tid];                                          \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 35] = num_ou;                                       \
    }                                                                       \
    if(offset ==0) {                                                        \
        newj[ii] = num_in;                                                  \
    }                                                                       \
	simdsync();


#define local_allocate_acc_dGij()                                           \
    __local numtyp red_accj[24][BLOCK_PAIR];								\
	__shared__ numtyp red_acck2[BLOCK_PAIR];

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkyL, dG_dkz, begin_k,            \
                 ntsf, tid, offset, t_per_atom, myMAX)                      \
    red_acck2[tid] = dG_dkyL;                                               \
    for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {                 \
        simdsync();                                                         \
        if(offset < s)                                                      \
            red_acck2[tid] += red_acck2[tid + s];                           \
    }                                                                       \
    for (int m = 0; m < ntsf; m++) {                                        \
        red_accj[m][tid] = dG_dkz[m];                                       \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                if(m != myMAX)                                              \
                     dG_dky[m][tid] += dG_dky[m][tid + s];                  \
                red_accj[m][tid] += red_accj[m][tid + s];                   \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            if(m != myMAX)                                                  \
                dGij[index_bm].y += dG_dky[m][tid];                         \
            dGij[index_bm].z += red_accj[m][tid];                           \
        }                                                                   \
    }                                                                       \
	if (offset == 0)                                                        \
		dGij[begin_k + myMAX].y += red_acck2[tid];

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    red_accG[tid] += red_accG[tid + s];                     \
                }                                                           \
            }                                                               \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }                                                                       \
    numtyp sfsc, sfmi;                                                      \
    for (int i = 0; i < nsf; i++) {                                         \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfmi, i, sfmi_tex);                                           \
        dG_dj[i].w = (dG_dj[i].w - sfmi) / sfsc;                            \
    }                                                                       \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            red_accG[tid] = red_accG[tid - offset];                         \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }

#else

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 35] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            num_in += shfl_down(num_in, s, t_per_atom);                     \
        }                                                                   \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 35] = num_ou;                                       \
    }                                                                       \
    if (offset == 0) {                                                      \
        newj[ii] = num_in;                                                  \
    }                                                                       \
	simdsync();

#define local_allocate_acc_dGij()

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkyL, dG_dkz, begin_k,            \
                 ntsf, tid, offset, t_per_atom, myMAX)                      \
    for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1)                   \
        dG_dkyL += shfl_down(dG_dkyL, s, t_per_atom);                       \
    for (int m = 0; m < ntsf; m++) {                                        \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            dG_dkz[m] += shfl_down(dG_dkz[m], s, t_per_atom);               \
            simdsync();                                                     \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                if(m != myMAX)                                              \
                    dG_dky[m][tid] += dG_dky[m][tid + s];                   \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            if(m != myMAX)                                                  \
                dGij[index_bm].y += dG_dky[m][tid];                         \
            dGij[index_bm].z += dG_dkz[m];                                  \
        }                                                                   \
    }                                                                       \
    if(offset == 0)                                                         \
       dGij[begin_k + myMAX].y += dG_dkyL;

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    numtyp sfsc, sfmi;                                                      \
    for (int i = 0; i < nsf; i++) {                                         \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfmi, i, sfmi_tex);                                           \
        dG_dj[i].w = (dG_dj[i].w - sfmi) / sfsc;                            \
    }

#endif

//---------------------------------------------------------------------
	// get the short neighbor list
//----------------------------------------------------------------------
__kernel void k_annp_short_nbor(const __global numtyp4* restrict x_,
								const numtyp cutoff,
								__global int* dev_nbor,
								__global int* dev_packed,
								__global int* newj, const int inum, 
								const int nbor_pitch, const int t_per_atom) {
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);
	local_allocate_acc_numj();

	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;																// for getting the information of the neighbor-list
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
				  offset, i, jnum, n_stride, nbor_end, nbor_j);

		numtyp4 ix; fetch4(ix, i, pos_tex);
		int nbor_begin = nbor_j;																			
		int index_in = 0;																		
		int index_ou = 0;
		int j_out[300], j_in[300];																
		
		numtyp cutoff2 = cutoff * cutoff;
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int sj = dev_packed[nbor_j];
			int sj_nomask = sj;
			sj &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, sj, pos_tex);

			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = CFLENGTH * CFLENGTH * (deltx * deltx + delty * delty + deltz * deltz);
			if (r2ij < cutoff2) {
				j_in[index_in] = sj_nomask;
				index_in++;
			}
			else {
				j_out[index_ou] = sj_nomask;
				index_ou++;
			}
		}																						
		int in_out[70];																				
		acc_numj(newj, in_out, ii, index_in, index_ou, tid, t_per_atom, offset);					
		int numj = newj[ii];
		dev_nbor[ii + nbor_pitch] = numj;													

		int sum_in = 0, sum_ou = numj;
		nbor_begin -= offset;																	
		for (int j = 0; j < offset; j++) {
			sum_in += in_out[j];
			sum_ou += in_out[j + 35];
		}
		for (int j = 0; j < in_out[offset]; j++) {
			int index = sum_in + j;
			int begin_in = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_in] = j_in[j];
		}
		for (int j = 0; j < in_out[offset + 35]; j++) {
			int index = sum_ou + j;
			int begin_ou = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_ou] = j_out[j];
		}
	}
}

//---------------------------------------------------------------------
	// __kernel extern "C" __global__, in "ucl_nv_kernel.h" file
//----------------------------------------------------------------------
__kernel void k_annp(const __global numtyp4* restrict x_, const int ntypes,
				 	 const int ntl, const int nhl, const int nnod,
					 const int nsf, const int npsf, const int ntsf,
					 const int t_per_atom, const __global numtyp* restrict sf_scal,
					 const __global numtyp* restrict sf_min,
					 const numtyp4 out_mod, const int eflag,
					 const __global numtyp* restrict weight_all,
					 const __global numtyp* restrict bias_all, const int inum,
					 const __global int* flagact, const __global int* dev_nbor,
					 const __global int* dev_packed, const int nbor_pitch,
					 __global acctyp4* Fj, __global numtyp4* dGij,
					 __global acctyp* restrict engv, 
					 __global acctyp4* restrict force, 
					 const __global numtyp4* restrict coeff_sym,
					 const int2 gpup, const int begin_i, const __global int* restrict newj) {

	int max_nbor_size = gpup.x;

	local_allocate_acc_dGij();
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);														

	//---------------- starting calculation
	int begin_jk = ii * max_nbor_size;														
	int index_bm;
	ii += begin_i;																				
	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
			      offset, i, jnum, n_stride, nbor_end, nbor_j);									

		numtyp4 ix; fetch4(ix, i, pos_tex);														
		int idj = offset;																		
		numtyp4 dG_dj[27] = { 0.0,0.0,0.0,0.0 };

		//-------------------- starting -------------
		for (; nbor_j < nbor_end; nbor_j += n_stride, idj += t_per_atom) {
			for (int k = 0; k < nsf; k++) {														
				dG_dj[k].x = (numtyp)0.0;
				dG_dj[k].y = (numtyp)0.0;
				dG_dj[k].z = (numtyp)0.0;
			}
			int j = dev_packed[nbor_j];													
			j &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, j, pos_tex);								
			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;
			numtyp rij = ucl_sqrt(r2ij);
			numtyp rij_m = rij * CFLENGTH;
			numtyp r2ij_m = rij_m * rij_m;

			numtyp4 dr_dj, coeff;														
			dr_dj.x = -deltx / rij;
			dr_dj.y = -delty / rij;
			dr_dj.z = -deltz / rij;
			numtyp Rc, coe_0, coe_fc, fcij, dfcij, term1, term2, term3;
			fetch4(coeff, 0, cofsym_tex);
			Rc = coeff.w;

			coe_0 = MY_PI / Rc;
			coe_fc = coe_0 * rij_m;
			fcij = 0.5 * (cos(coe_fc) + 1.0);
			dfcij = -0.5 * coe_0 * sin(coe_fc);
			for (int m = 0; m < npsf; m++) {
				fetch4(coeff, m, cofsym_tex);
				term1 = ucl_exp(-coeff.x * r2ij_m);
				term2 = term1 * (-2.0 * fcij * coeff.x * rij_m + dfcij);				
				dG_dj[m].w += term1 * fcij;													
				dG_dj[m].x += term2 * dr_dj.x;		
				dG_dj[m].y += term2 * dr_dj.y;												
				dG_dj[m].z += term2 * dr_dj.z;
			}

			//-------------- triple symmetry function
			int idk, nbor_k; 
			numtyp4 dr_dk, dr_djk, dct_dj, dct_dk, term2_drj;
			numtyp4 term2_drk, term2_t, term3_drj, term3_drk;

			nbor_k = nbor_j + n_stride;
			idk = idj + 1;
			for (; nbor_k < nbor_end; nbor_k += n_stride, idk++) {
				//if (nbor_k == nbor_j)	continue;			
				numtyp dG_dkx[24] = { 0.0 };
				numtyp dG_dky[24] = { 0.0 };
				numtyp dG_dkz[24] = { 0.0 };

				int k = dev_packed[nbor_k];
				k &= NEIGHMASK;
				numtyp4 kx; fetch4(kx, k, pos_tex);

				numtyp delt2x = ix.x - kx.x;								numtyp delt2y = ix.y - kx.y;
				numtyp delt2z = ix.z - kx.z;
				numtyp delt3x = jx.x - kx.x;								numtyp delt3y = jx.y - kx.y;
				numtyp delt3z = jx.z - kx.z;
				numtyp r2ik = delt2x * delt2x + delt2y * delt2y + delt2z * delt2z;
				numtyp r2jk = delt3x * delt3x + delt3y * delt3y + delt3z * delt3z;
				numtyp rik = ucl_sqrt(r2ik);
				numtyp rjk = ucl_sqrt(r2jk);
				numtyp rik_m = rik * CFLENGTH;
				numtyp rjk_m = rjk * CFLENGTH;

				if (rjk_m < Rc) {
					numtyp rinv12 = ucl_recip(rij * rik);
					numtyp cos_theta = (deltx * delt2x + delty * delt2y + deltz * delt2z) * rinv12;
					dr_dk.x = -delt2x / rik;								dr_dk.y = -delt2y / rik;
					dr_dk.z = -delt2z / rik;
					dr_djk.x = delt3x / rjk;								dr_djk.y = delt3y / rjk;
					dr_djk.z = delt3z / rjk;

					term1 = cos_theta / r2ij;
					term2 = cos_theta / r2ik;
					dct_dj.x = -delt2x * rinv12 + term1 * deltx;
					dct_dj.y = -delt2y * rinv12 + term1 * delty;
					dct_dj.z = -delt2z * rinv12 + term1 * deltz;
					dct_dk.x = -deltx * rinv12 + term2 * delt2x;
					dct_dk.y = -delty * rinv12 + term2 * delt2y;
					dct_dk.z = -deltz * rinv12 + term2 * delt2z;					

					numtyp r2sum = CFLENGTH * CFLENGTH * (r2ij + r2ik + r2jk);
					term2_t.x = rjk_m * dr_djk.x;							term2_t.y = rjk_m * dr_djk.y;
					term2_t.z = rjk_m * dr_djk.z;
					term2_drj.x = 2.0 * (rij_m * dr_dj.x + term2_t.x);		term2_drj.y = 2.0 * (rij_m * dr_dj.y + term2_t.y);
					term2_drj.z = 2.0 * (rij_m * dr_dj.z + term2_t.z);
					term2_drk.x = 2.0 * (rik_m * dr_dk.x - term2_t.x);		term2_drk.y = 2.0 * (rik_m * dr_dk.y - term2_t.y);
					term2_drk.z = 2.0 * (rik_m * dr_dk.z - term2_t.z);

					numtyp coe_fcik = coe_0 * rik_m;
					numtyp fcik = 0.5 * (cos(coe_fcik) + 1.0);
					numtyp dfcik = -0.5 * coe_0 * sin(coe_fcik);

					numtyp coe_fcjk = coe_0 * rjk_m;
					numtyp fcjk = 0.5 * (cos(coe_fcjk) + 1.0);
					numtyp dfcjk = -0.5 * coe_0 * sin(coe_fcjk);

					numtyp term_fc = fcij * fcik * fcjk;
					numtyp4 term3_t;
					term3_t.x = fcjk * dfcij;								term3_t.y = fcij * dfcjk;
					term3_t.z = fcjk * dfcik;								term3_t.w = fcik * dfcjk;
					term3_drj.x = fcik * (term3_t.x * dr_dj.x + term3_t.y * dr_djk.x);
					term3_drj.y = fcik * (term3_t.x * dr_dj.y + term3_t.y * dr_djk.y);
					term3_drj.z = fcik * (term3_t.x * dr_dj.z + term3_t.y * dr_djk.z);
					term3_drk.x = fcij * (term3_t.z * dr_dk.x - term3_t.w * dr_djk.x);
					term3_drk.y = fcij * (term3_t.z * dr_dk.y - term3_t.w * dr_djk.y);
					term3_drk.z = fcij * (term3_t.z * dr_dk.z - term3_t.w * dr_djk.z);

					for (int m = 0; m < ntsf; m++) {
						int index_t = npsf + m;
						fetch4(coeff, index_t, cofsym_tex);											// eta, lambda, zeta, Rc (x, y, z, w)
						numtyp flag = 1 + coeff.y * cos_theta;
						if (flag <= 0)	continue;

						numtyp term_coe = pow(2, 1 - coeff.z);
						numtyp term_cot = term_coe * pow(flag, coeff.z);
						numtyp term_exp = ucl_exp(-coeff.x * r2sum);
						dG_dj[index_t].w += term_cot * term_exp * term_fc;

						term1 = coeff.y * term_cot * term_exp * term_fc * coeff.z / flag / CFLENGTH;
						term3 = term_cot * term_exp;
						term2 = term3 * term_fc * coeff.x;		
						dG_dj[index_t].x += term1 * dct_dj.x - term2 * term2_drj.x + term3 * term3_drj.x;
						dG_dj[index_t].y += term1 * dct_dj.y - term2 * term2_drj.y + term3 * term3_drj.y;
						dG_dj[index_t].z += term1 * dct_dj.z - term2 * term2_drj.z + term3 * term3_drj.z;
						dG_dkx[m] = term1 * dct_dk.x - term2 * term2_drk.x + term3 * term3_drk.x;
						dG_dky[m] = term1 * dct_dk.y - term2 * term2_drk.y + term3 * term3_drk.y;
						dG_dkz[m] = term1 * dct_dk.z - term2 * term2_drk.z + term3 * term3_drk.z;
					}
				}
				// update the value of dG_dk
				int begin_k = (begin_jk + idk) * nsf + npsf;
				for (int m = 0; m < ntsf; m++) {
					index_bm = begin_k + m;
					dGij[index_bm].x += dG_dkx[m];
					dGij[index_bm].y += dG_dky[m];
					dGij[index_bm].z += dG_dkz[m];
				}
			}

			int begin_j = (begin_jk + idj) * nsf;
			dGij[begin_j].w = (numtyp)j;
			for (int m = 0; m < nsf; m++) {
				index_bm = begin_j + m;
				dGij[index_bm].x += dG_dj[m].x;
				dGij[index_bm].y += dG_dj[m].y;
				dGij[index_bm].z += dG_dj[m].z;
			} 
		}	// j loop
		acc_Gi(dG_dj, sf_scal, sf_min, nsf, tid, t_per_atom, offset);

		numtyp weight, bias, tsf_scal;
		numtyp hidly[24] = { 0.0 };															
		numtyp t_hidly[24] = { 0.0 };
		numtyp hidly_d[24] = { 0.0 };														
		numtyp lays_dw[24 * 27] = { 0.0 };													
		numtyp temp_dw[24 * 27] = { 0.0 };
		numtyp hidly_dw[24 * 27] = { 0.0 };																								
		int index_w = 0, index_w2 = 0, index_t;
		int2 nrc_w[3];
		nrc_w[0].x = nnod; nrc_w[0].y = nsf;
		nrc_w[1].x = nnod; nrc_w[1].y = nnod;
		nrc_w[2].x = 1; nrc_w[2].y = nnod;
		for (int n = 0; n < ntl - 1; n++) {
			for (int k = 0; k < nnod; k++) {
				t_hidly[k] = 0.0;
				index_t = k * nsf;
				for (int m = 0; m < nsf; m++) {													
					lays_dw[index_t + m] = 0.0;
				}
			}
			int actflag = flagact[n];
			for (int k = 0; k < nrc_w[n].x; k++) {
				fetch(bias, k + n * nnod, bias_tex);
				for (int m = 0; m < nrc_w[n].y; m++) {
					//fetch(weight, index_w, weight_tex);
					weight = weight_all[index_w];
					if (n == 0) {
						t_hidly[k] += weight * dG_dj[m].w;
					}
					else {
						t_hidly[k] += weight * hidly[m];
					}
					index_w++;
				}
				t_hidly[k] += bias;
			}
			for (int k = 0; k < nrc_w[n].x; k++) {												
				if (actflag == 0) {
					hidly[k] = t_hidly[k];
					hidly_d[k] = 1;
				}
				if (actflag == 4) {
					numtyp t_exp = t_hidly[k];
					hidly[k] = (ucl_exp(t_exp) - ucl_exp(-t_exp)) / (ucl_exp(t_exp) + ucl_exp(-t_exp));
					hidly_d[k] = (1.0 - hidly[k] * hidly[k]);
				}
			}
			// for the dE_dG
			int index_dw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {											
				for (int m = 0; m < nrc_w[n].y; m++) {
					//fetch(weight, index_w2, weight_tex);											// cannot be used for RTX A5000
					weight = weight_all[index_w2];
					hidly_dw[index_dw] = hidly_d[k] * weight;
					index_w2++;
					index_dw++;
				}
			}																					
			int index_tdw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {
				for (int m = 0; m < nsf; m++) {
					if (n == 0) {
						temp_dw[k * nsf + m] = hidly_dw[k * nsf + m];
					}
					else {
						for (int j = 0; j < nrc_w[n].y; j++) {
							lays_dw[index_tdw] += hidly_dw[k * nnod + j] * temp_dw[j * nsf + m];
						}
						index_tdw++;
					}
				}
			}
			if (n != 0 && n != ntl - 1)															
				for (int k = 0; k < nrc_w[n].x; k++) {
					for (int m = 0; m < nsf; m++) {
						temp_dw[k * nsf + m] = lays_dw[k * nsf + m];
					}
				}																			
		}
		engv[ii] = hidly[0];																		// updating the erergy

		// force for atom i and j
		acctyp4 F;
		int begin_i = begin_jk + ii;
		for (int jj = offset; jj < jnum; jj += t_per_atom) {
			F.x = 0.0; F.y = 0.0; F.z = 0.0;
			int begin_j = (begin_jk + jj) * nsf;
			for (int k = 0; k < nsf; k++) {
				fetch(tsf_scal, k, sfsc_tex);
				F.x -= lays_dw[k] * dGij[begin_j + k].x / tsf_scal * CFFORCE;
				F.y -= lays_dw[k] * dGij[begin_j + k].y / tsf_scal * CFFORCE;
				F.z -= lays_dw[k] * dGij[begin_j + k].z / tsf_scal * CFFORCE;
			}
			int index_jk = begin_i + jj + 1;
			Fj[index_jk].x += F.x;
			Fj[index_jk].y += F.y;
			Fj[index_jk].z += F.z;
			Fj[index_jk].w = dGij[begin_j].w;

			Fj[begin_i].x -= F.x;
			Fj[begin_i].y -= F.y;
			Fj[begin_i].z -= F.z;
		}
	}	// if ii
}

//----------------------------------------------------------------------
	// updating the force for neighbor
//----------------------------------------------------------------------
__kernel void k_annp_updat(const __global numtyp4* restrict x_,
						   const __global int* restrict newj, 
						   const __global acctyp4* restrict Fj,
						   __global acctyp4* force,
						   __global acctyp2* virial2,
						   __global acctyp4* virial4,
						   const int eflag, const int vflag, 
						   const int begin_i, const int2 gpup) {

	int max_nbor_size = gpup.x;
	int num_atoms = gpup.y;
	int tid = THREAD_ID_X;
	__shared__ int ii;
	__shared__ numtyp4 ix;

	acctyp2 old_v2;
	acctyp4 old_v4;
	for (ii = 0; ii < num_atoms; ) {
		__shared__ acctyp4 tFj[BLOCK_PAIR];
		__shared__ acctyp tvirial[BLOCK_PAIR][6];

		int indexi = ii + begin_i;
		int n_jnum = newj[indexi];
		int begin_jk = ii * max_nbor_size;
		int begin_i = begin_jk + ii;
		fetch4(ix, indexi, pos_tex);															
		
		if (tid == 0) {																		
			acctyp4 old_f = force[indexi];
			old_f.x += Fj[begin_i].x;
			old_f.y += Fj[begin_i].y;
			old_f.z += Fj[begin_i].z;
			force[indexi] = old_f;
		}
		if (tid < n_jnum) {
			int idj = begin_i + tid + 1;
			int indexj = (int)Fj[idj].w;
			tFj[tid].x = Fj[idj].x;
			tFj[tid].y = Fj[idj].y;
			tFj[tid].z = Fj[idj].z;

			acctyp4 old_f = force[indexj];												
			old_f.w = indexj;
			old_f.x += tFj[tid].x;
			old_f.y += tFj[tid].y;
			old_f.z += tFj[tid].z;
			force[indexj] = old_f;

			if(EVFLAG && vflag) {
				numtyp4 jx; fetch4(jx, indexj, pos_tex);
				numtyp delx = ix.x - jx.x;
				numtyp dely = ix.y - jx.y;
				numtyp delz = ix.z - jx.z;

				tvirial[tid][0] = delx*-tFj[tid].x;
				tvirial[tid][1] = dely*-tFj[tid].y;
				tvirial[tid][2] = delz*-tFj[tid].z;
				tvirial[tid][3] = delx*-tFj[tid].y;
				tvirial[tid][4] = delx*-tFj[tid].z;
			
				old_v2 = virial2[indexj];
				old_v4 = virial4[indexj];	
				old_v4.x += 0.5 * tvirial[tid][0];
				old_v4.y += 0.5 * tvirial[tid][1];
				old_v4.z += 0.5 * tvirial[tid][2];
				old_v4.w += 0.5 * tvirial[tid][3];
				old_v2.x += 0.5 * tvirial[tid][4];
				old_v2.y += 0.5 * tvirial[tid][5];
				virial2[indexj] = old_v2;
				virial4[indexj] = old_v4;
			}
		}
		if (EVFLAG && vflag) {
			for (unsigned int s = n_jnum / 2; s > 0; s >>= 1) {
				int idtid = tid + s;
				__syncthreads();
				if (tid < s) {
					tvirial[tid][0] += tvirial[idtid][0];
					tvirial[tid][1] += tvirial[idtid][1];
					tvirial[tid][2] += tvirial[idtid][2];
					tvirial[tid][3] += tvirial[idtid][3];
					tvirial[tid][4] += tvirial[idtid][4];
					tvirial[tid][5] += tvirial[idtid][5];
				}
				__syncthreads();
				if (s % 2 == 1 && s != 1 && tid == 0) {											
					idtid -= 1;
					tvirial[tid][0] += tvirial[idtid][0];
					tvirial[tid][1] += tvirial[idtid][1];
					tvirial[tid][2] += tvirial[idtid][2];
					tvirial[tid][3] += tvirial[idtid][3];
					tvirial[tid][4] += tvirial[idtid][4];
					tvirial[tid][5] += tvirial[idtid][5];
				}
			}
			if (n_jnum % 2 == 1 && tid == 0) {												
				int idtid = tid + n_jnum - 1;
				tvirial[tid][0] += tvirial[idtid][0];
				tvirial[tid][1] += tvirial[idtid][1];
				tvirial[tid][2] += tvirial[idtid][2];
				tvirial[tid][3] += tvirial[idtid][3];
				tvirial[tid][4] += tvirial[idtid][4];
				tvirial[tid][5] += tvirial[idtid][5];
			}
			if (tid == 0) {
				old_v2 = virial2[indexi];
				old_v4 = virial4[indexi];						
				old_v4.x += 0.5 * tvirial[tid][0];
				old_v4.y += 0.5 * tvirial[tid][1];
				old_v4.z += 0.5 * tvirial[tid][2];
				old_v4.w += 0.5 * tvirial[tid][3];
				old_v2.x += 0.5 * tvirial[tid][4];
				old_v2.y += 0.5 * tvirial[tid][5];
				virial2[indexi] = old_v2;
				virial4[indexi] = old_v4;
			}
		}
		__syncthreads();
		ii++;
	}
}
