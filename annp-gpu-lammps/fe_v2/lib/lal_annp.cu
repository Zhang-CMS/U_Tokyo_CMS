#include "hip/hip_runtime.h"
//* Device code *---------------------------------------
//      Artifical Neural Network Potential
//             Accelerated by GPU
//______________________________________________________        
//  begin:  Wed February 16, 2022
//  email:  
//______________________________________________________
//------------------------------------------------------

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"

#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( weight_tex,float);
_texture( bias_tex,float);
_texture( sfsc_tex,float);
_texture( sfav_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( weight_tex,int2);
_texture( bias_tex,int2);
_texture( sfsc_tex,int2);
_texture( sfav_tex,int2);
#endif

#if (__CUDACC_VER_MAJOR__ >= 11)
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sfnor_scal
#define sfav_tex sfnor_avg
#endif

#else
#define pos_tex x_
#define weight_tex weight_all;
#define bias_tex bias_all
#define sfsc_tex sfnor_scal
#define sfav_tex sfnor_avg
#endif

#define MY_PI (numtyp)3.14159265358979323846
#define coeff_a (numtyp)1.7159
#define coeff_b (numtyp)0.666666666666667
#define coeff_c (numtyp)0.1

#if (SHUFFLE_AVAIL == 0)

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 20] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if (offset < s) {                                               \
                red_accj_in[tid] += red_accj_in[tid + s];                   \
            }                                                               \
        }                                                                   \
        num_in = red_accj_in[tid];                                          \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 20] = num_ou;                                       \
    }                                                                       \
    if(offset == 0) {                                                       \
        newj[ii] = num_in;                                                  \
    }                                                                       \
	simdsync();


#define local_allocate_acc_dGij()                                           \
    __local numtyp red_accj[19][BLOCK_PAIR];

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k,                     \
                 ntsf, tid, offset, t_per_atom)                             \
    for (int m = 0; m < ntsf; m++) {                                        \
        red_accj[m][tid] = dG_dk[m].z;                                      \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            simdsync();                                                     \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                dG_dky[m][tid] += dG_dky[m][tid + s];                       \
                red_accj[m][tid] += red_accj[m][tid + s];                   \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            dGij[index_bm].y += dG_dky[m][tid];                             \
            dGij[index_bm].z += red_accj[m][tid];                           \
        }                                                                   \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    red_accG[tid] += red_accG[tid + s];                     \
                }                                                           \
            }                                                               \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }                                                                       \
    numtyp sfsc, sfav;                                                      \
    for (int i = 0; i < nsf; i++) {                                         \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfav, i, sfav_tex);                                           \
        dG_dj[i].w = sfsc * dG_dj[i].w - sfsc * sfav;                       \
    }                                                                       \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            red_accG[tid] = red_accG[tid - offset];                         \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }

#define local_allocate_acc_Fi()                                             \
    __local acctyp red_accfi[3][BLOCK_PAIR];

#define acc_Fi(Fi, force, i, tid, t_per_atom, offset)                       \
    if (t_per_atom > 1) {                                                   \
        red_accfi[0][tid] = Fi.x;                                           \
        red_accfi[1][tid] = Fi.y;                                           \
        red_accfi[2][tid] = Fi.z;                                           \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
                simdsync();                                                 \
                if (offset < s) {                                           \
                    for (int i = 0; i < 3; i++)                             \
                        red_accfi[i][tid] += red_accfi[i][tid + s];         \
                }                                                           \
        }                                                                   \
    }                                                                       \
    if (offset == 0 && ii) {                                                \
        acctyp4 old_f = force[ii];                                          \
        old_f.x += red_accfi[0][tid];                                       \
        old_f.y += red_accfi[1][tid];                                       \
        old_f.z += red_accfi[2][tid];                                       \
        force[ii] = old_f;                                                  \
    }

#else

#define local_allocate_acc_numj()                                           \
    __local int red_accj_in[BLOCK_PAIR];                                    \
    __local int red_accj_ou[BLOCK_PAIR];

#define acc_numj(newj, in_out, ii, num_in, num_ou, tid, t_per_atom, offset) \
    if (t_per_atom > 1) {                                                   \
        red_accj_in[tid] = num_in;                                          \
        red_accj_ou[tid] = num_ou;                                          \
        for (int s = 0; s < t_per_atom; s++) {                              \
            in_out[s] = red_accj_in[tid - offset + s];                      \
            in_out[s + 20] = red_accj_ou[tid - offset + s];                 \
        }                                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            num_in += shfl_down(num_in, s, t_per_atom);                     \
        }                                                                   \
    }                                                                       \
    else {                                                                  \
        in_out[offset] = num_in;                                            \
        in_out[offset + 20] = num_ou;                                       \
    }                                                                       \
    if (offset == 0) {                                                      \
        newj[ii] = num_in;                                                  \
    }                                                                       \
	simdsync();

#define local_allocate_acc_dGij()

#define acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k,                     \
                 ntsf, tid, offset, t_per_atom)                             \
    for (int m = 0; m < ntsf; m++) {                                        \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            dG_dkz[m] += shfl_down(dG_dkz[m], s, t_per_atom);               \
            if(offset < s) {                                                \
                dG_dkx[m][tid] += dG_dkx[m][tid + s];                       \
                dG_dky[m][tid] += dG_dky[m][tid + s];                       \
            }                                                               \
        }                                                                   \
        if(offset == 0) {                                                   \
            int index_bm = begin_k + m;                                     \
            dGij[index_bm].x += dG_dkx[m][tid];                             \
            dGij[index_bm].y += dG_dky[m][tid];                             \
            dGij[index_bm].z += dG_dkz[m];                                  \
        }                                                                   \
    }

#define local_allocate_acc_Gi()                                             \
    __local numtyp red_accG[BLOCK_PAIR];

#define acc_Gi(dG_dj, sf_scal, sf_avg, nsf, tid, t_per_atom, offset)        \
    if (t_per_atom > 1) {                                                   \
       for (int m = 0; m < nsf; m++) {	                                    \
            for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {         \
                dG_dj[m].w += shfl_down(dG_dj[m].w, s, t_per_atom);         \
            }                                                               \
        }                                                                   \
    }                                                                       \
    numtyp sfsc, sfav;                                                      \
    for(int i = 0; i < nsf; i++) {                                          \
        fetch(sfsc, i, sfsc_tex);                                           \
        fetch(sfav, i, sfav_tex);                                           \
        dG_dj[i].w = sfsc * dG_dj[i].w - sfsc * sfav;                       \
    }                                                                       \
    if (t_per_atom > 1) {                                                   \
        for (int i = 0; i < nsf; i++) {                                     \
            red_accG[tid] = dG_dj[i].w;                                     \
            red_accG[tid] = red_accG[tid - offset];                         \
            dG_dj[i].w = red_accG[tid];                                     \
        }                                                                   \
    }

#define local_allocate_acc_Fi()

#define acc_Fi(Fi, force, ii, tid, t_per_atom, offset)                      \
    if (t_per_atom > 1) {                                                   \
        for (unsigned int s = t_per_atom / 2; s > 0; s >>= 1) {             \
            Fi.x += shfl_down(Fi.x, s, t_per_atom);                         \
            Fi.y += shfl_down(Fi.y, s, t_per_atom);                         \
            Fi.z += shfl_down(Fi.z, s, t_per_atom);                         \
        }                                                                   \
    }                                                                       \
    if (offset == 0) {                                                      \
        acctyp4 old_f = force[ii];                                          \
        old_f.x += Fi.x;                                                    \
        old_f.y += Fi.y;                                                    \
        old_f.z += Fi.z;                                                    \
        force[ii] = old_f;                                                  \
    }

#endif

//---------------------------------------------------------------------
	// get the short neighbor list
//----------------------------------------------------------------------
__kernel void k_annp_short_nbor(const __global numtyp4* restrict x_,
								const numtyp cutMax,
								const int ntypes, 
								__global int* dev_nbor,
								__global int* dev_packed,
								__global int* newj, const int inum, 
								const int nbor_pitch, const int t_per_atom) {
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);

	local_allocate_acc_numj();
	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;																// get the information of the neighbor-list
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
				  offset, i, jnum, n_stride, nbor_end, nbor_j);

		numtyp4 ix; fetch4(ix, i, pos_tex);
		int nbor_begin = nbor_j;																			
		int index_in = 0;																			// for accumulating the number of j
		int index_ou = 0;
		int j_out[300], j_in[300];																	// j_in save for neighbor j within cutoff, j_out save for j without  cutoff
		
		for (; nbor_j < nbor_end; nbor_j += n_stride) {
			int sj = dev_packed[nbor_j];
			int sj_nomask = sj;
			sj &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, sj, pos_tex);

			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;

			if (r2ij <= cutMax && r2ij > 1.0e-10) {
				j_in[index_in] = sj_nomask;
				index_in++;
			}
			else {
				j_out[index_ou] = sj_nomask;
				index_ou++;
			}
		}																							// cutsq is modified in "lal_atom.h" file, change it to one array			
		int in_out[40];																				// for saving the number of "in" vlaue (0-9), and "out" value(10-19)�� maximum value is 10 > t_per_atoms
		acc_numj(newj, in_out, ii, index_in, index_ou, tid, t_per_atom, offset);					// accumulate the number of "J" atoms, the value saved into "newj" matirx
		int numj = newj[ii];
		dev_nbor[ii + nbor_pitch] = numj;															// update the number of neighbors

		int sum_in = 0, sum_ou = numj;
		nbor_begin -= offset;																		// to initial position
		for (int j = 0; j < offset; j++) {
			sum_in += in_out[j];
			sum_ou += in_out[j + 20];
		}
		for (int j = 0; j < in_out[offset]; j++) {
			int index = sum_in + j;
			int begin_in = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_in] = j_in[j];
		}
		for (int j = 0; j < in_out[offset + 20]; j++) {
			int index = sum_ou + j;
			int begin_ou = nbor_begin + (index / t_per_atom) * n_stride + index % t_per_atom;
			dev_packed[begin_ou] = j_out[j];
		}
	}
}

//---------------------------------------------------------------------
	// __kernel extern "C" __global__, in "ucl_nv_kernel.h" file
//----------------------------------------------------------------------
__kernel void k_annp(const __global numtyp4* restrict x_, const int ntypes,
				 	 const int ntl, const int nhl, const int nnod,
					 const int nsf, const int npsf, const int ntsf,
					 const __global int* restrict map, const int t_per_atom,
					 const numtyp cutMax,
					 const __global numtyp* restrict sfnor_scal,
					 const __global numtyp* restrict sfnor_avg,
					 const numtyp4 out_mod, const int eflag,
					 const __global numtyp* restrict weight_all,
					 const __global numtyp* restrict bias_all, const int inum,
					 const __global int* flagact, const __global int* dev_nbor,
					 const __global int* dev_packed, const int nbor_pitch,
					 __global acctyp4* Fj, 
					 __global acctyp* restrict engv, 
					 __global acctyp4* restrict force, 
					 __global numtyp4* dGij,
					 const int2 gpup, const int begin_i, const __global int* restrict newj) {

	int max_nbor_size = gpup.x;
	numtyp e_scale = out_mod.x;																		// will be changed into numtyp format
	numtyp e_shift = out_mod.y;
	numtyp e_atom = out_mod.z;
	numtyp Rc = ucl_sqrt(cutMax);

	local_allocate_acc_dGij();
	local_allocate_acc_Gi();
	local_allocate_acc_Fi();
	int tid, ii, offset, n_stride;
	atom_info(t_per_atom, ii, tid, offset);															// in "lal_aux_fun1.h" file	

	//---------------- starting calculation
	int begin_jk = ii * max_nbor_size;																// Fj, dGij, matrix always start from 0
	int index_bm;
	ii += begin_i;																					// ii should update

	//if(ii == 2)
	if (ii < inum) {
		int i, nbor_j, nbor_end, jnum;
		nbor_info(dev_nbor, dev_packed, nbor_pitch, t_per_atom, ii, 
			      offset, i, jnum, n_stride, nbor_end, nbor_j);										// get the nbor information of atom i

		numtyp4 ix; fetch4(ix, i, pos_tex);															// define in "lal_preporcessor.h" file
		int idj = offset;																			// index for dGij, because the 0-28 is for atom i
		numtyp4 dG_dj[28] = { 0.0,0.0,0.0,0.0 };

		numtyp dG_dkz[19];																			// for saving the value of dG_dk, just can two shared memory fo dG_dkx, dG_dky
		__shared__ numtyp dG_dkx[19][BLOCK_PAIR];
		__shared__ numtyp dG_dky[19][BLOCK_PAIR];

		for (; nbor_j < nbor_end; nbor_j += n_stride, idj += t_per_atom) {
			for (int k = 0; k < nsf; k++) {															// set zero for atom j
				dG_dj[k].x = (numtyp)0.0;
				dG_dj[k].y = (numtyp)0.0;
				dG_dj[k].z = (numtyp)0.0;
			}
			int j = dev_packed[nbor_j];																// the 3rd row is the starting location in "packed nobrs"
			j &= NEIGHMASK;
			numtyp4 jx; fetch4(jx, j, pos_tex);														// get the coordinates of j
			numtyp deltx = ix.x - jx.x;
			numtyp delty = ix.y - jx.y;
			numtyp deltz = ix.z - jx.z;
			numtyp r2ij = deltx * deltx + delty * delty + deltz * deltz;

			//---------------- pair symmetry function	
			numtyp x_fc, xij, term1, term2, term1_coeff, term2_coeff;
			numtyp rij = ucl_sqrt(r2ij);
			numtyp coe_0 = MY_PI / Rc;
			x_fc = coe_0 * rij;
			numtyp fcij = 0.5 * (cos(x_fc) + 1.0);
			numtyp dfcij = -0.5 * coe_0 * sin(x_fc);

			numtyp4 dr_dj, tx, dtx;
			dr_dj.x = -deltx / rij;
			dr_dj.y = -delty / rij;
			dr_dj.z = -deltz / rij;

			xij = 2.0 * rij / Rc - 1.0;
			tx.x = (numtyp)1.0;						tx.y = xij;
			dtx.x = (numtyp)0.0;					dtx.y = (numtyp)1.0;
			term1_coeff = 2.0 * fcij / Rc;

			dG_dj[0].w += fcij;
			dG_dj[0].x += dfcij * dr_dj.x;			dG_dj[0].y += dfcij* dr_dj.y;
			dG_dj[0].z += dfcij * dr_dj.z;

			dG_dj[1].w += fcij * xij;
			term1 = term1_coeff + xij * dfcij ;
			dG_dj[1].x += term1 * dr_dj.x;			dG_dj[1].y += term1 * dr_dj.y;
			dG_dj[1].z += term1 * dr_dj.z;

			for (int m = 2; m < npsf; m++) {
				tx.z = 2.0 * xij * tx.y - tx.x;
				dtx.z = 2.0 * tx.y + 2.0 * xij * dtx.y - dtx.x;
				tx.x = tx.y;						dtx.x = dtx.y;
				tx.y = tx.z;						dtx.y = dtx.z;

				dG_dj[m].w += fcij * tx.z;												// This is G value 
				term1 = dtx.z * term1_coeff + tx.z * dfcij;
				dG_dj[m].x += term1 * dr_dj.x;		dG_dj[m].y += term1 * dr_dj.y;
				dG_dj[m].z += term1 * dr_dj.z;
				//printf("G pair.... %d %d %d %d %f %f %f %f\n", m, ii, offset, j, rij, Rc, fcij, tx.z);
			}
			//-------------- triple symmetry function
			int idk = idj, nbor_k, nbor_kend, k_loop;
			k_loop = 1 + jnum / t_per_atom - (int)idj / t_per_atom;
			nbor_k = nbor_j;
			nbor_kend = nbor_j + t_per_atom - offset;
			for (int n = 0; n < k_loop; n++) {
				if (n != 0) {
					nbor_k = nbor_j;
					nbor_k += (n * n_stride - offset);
					nbor_kend = nbor_k + t_per_atom;
				}
				if (nbor_kend > nbor_end)	nbor_kend = nbor_end;
				for (; nbor_k < nbor_kend; nbor_k++, idk++) {
					if (nbor_k == nbor_j)	continue;
					for (int m = 0; m < ntsf; m++) {													// set zero for atom k
						dG_dkx[m][tid] = (numtyp)0.0;
						dG_dky[m][tid] = (numtyp)0.0;
						dG_dkz[m] = (numtyp)0.0;
					}

					int k = dev_packed[nbor_k];
					k &= NEIGHMASK;
					numtyp4 kx; fetch4(kx, k, pos_tex);
					numtyp delt2x = ix.x - kx.x;
					numtyp delt2y = ix.y - kx.y;
					numtyp delt2z = ix.z - kx.z;
					numtyp r2ik = delt2x * delt2x + delt2y * delt2y + delt2z * delt2z;
					numtyp rik = ucl_sqrt(r2ik);
					numtyp rinv12 = ucl_recip(rij * rik);
					numtyp cos_theta = (deltx * delt2x + delty * delt2y + deltz * delt2z) * rinv12;
					// if(ii == 2)
					// 	printf("check11... %d %d %d %d %d %f\n", ii, offset, j, k, n, dG_dj[9].w);
						
					x_fc = coe_0 * rik;
					numtyp fcik = 0.5 * (cos(x_fc) + 1.0);
					numtyp dfcik = -0.5 * coe_0 * sin(x_fc);

					numtyp4 dr_dk, dct_dj, dct_dk, tdGt_dj, tdGt_dk;
					numtyp term_cos1, term_cos2;
					dr_dk.x = - delt2x / rik;
					dr_dk.y = - delt2y / rik;
					dr_dk.z = - delt2z / rik;
					
					term_cos1 = cos_theta / r2ij;
					term_cos2 = cos_theta / r2ik;
					dct_dj.x = -delt2x * rinv12 + term_cos1 * deltx;
					dct_dj.y = -delt2y * rinv12 + term_cos1 * delty;
					dct_dj.z = -delt2z * rinv12 + term_cos1 * deltz;
					dct_dk.x = -deltx * rinv12 + term_cos2 * delt2x;
					dct_dk.y = -delty * rinv12 + term_cos2 * delt2y;
					dct_dk.z = -deltz * rinv12 + term_cos2 * delt2z;

					numtyp xik = 0.5 * (cos_theta + 1.0);
					tx.x = (numtyp)1.0;							tx.y = xik;
					dtx.x = (numtyp)0.0;						dtx.y = (numtyp)1.0;

					numtyp4 term2_dj, term2_dk;
					term2 = fcij * fcik;
					term2_coeff = dfcij * fcik;
					term2_dj.x = term2_coeff * dr_dj.x;			term2_dj.y = term2_coeff * dr_dj.y;
					term2_dj.z = term2_coeff * dr_dj.z;
					term2_coeff = dfcik * fcij;
					term2_dk.x = term2_coeff * dr_dk.x;			term2_dk.y = term2_coeff * dr_dk.y;
					term2_dk.z = term2_coeff * dr_dk.z;

					// for G0 value
					dG_dj[npsf].w += term2;
					dG_dj[npsf].x += term2_dj.x;				dG_dj[npsf].y += term2_dj.y;
					dG_dj[npsf].z += term2_dj.z;
					dG_dkx[0][tid] += term2_dk.x;				dG_dky[0][tid] += term2_dk.y;
					dG_dkz[0] += term2_dk.z;	

					// for G1 value
					int index_t = npsf + 1;
					dG_dj[index_t].w += tx.y * term2;
					term1_coeff = 0.5  * term2;
					tdGt_dj.x = term1_coeff * dct_dj.x + tx.y * term2_dj.x;
					tdGt_dj.y = term1_coeff * dct_dj.y + tx.y * term2_dj.y;
					tdGt_dj.z = term1_coeff * dct_dj.z + tx.y * term2_dj.z;
					tdGt_dk.x = term1_coeff * dct_dk.x + tx.y * term2_dk.x;
					tdGt_dk.y = term1_coeff * dct_dk.y + tx.y * term2_dk.y;
					tdGt_dk.z = term1_coeff * dct_dk.z + tx.y * term2_dk.z;

					dG_dj[index_t].x += tdGt_dj.x;				dG_dj[index_t].y += tdGt_dj.y;
					dG_dj[index_t].z += tdGt_dj.z;
					dG_dkx[1][tid] += tdGt_dk.x;				dG_dky[1][tid] += tdGt_dk.y;
					dG_dkz[1] += tdGt_dk.z;
					
					// for G2-ntsf value
					for (int m = 2; m < ntsf; m++) {
						index_t = npsf + m;
						tx.z = 2.0 * xik * tx.y - tx.x;
						dtx.z = 2.0 * tx.y + 2.0 * xik * dtx.y - dtx.x;
						tx.x = tx.y;
						dtx.x = dtx.y;
						tx.y = tx.z;
						dtx.y = dtx.z;
						dG_dj[index_t].w += tx.z * term2;

						numtyp t_term1_coeff = term1_coeff * dtx.z;
						tdGt_dj.x = t_term1_coeff * dct_dj.x + tx.z * term2_dj.x;
						tdGt_dj.y = t_term1_coeff * dct_dj.y + tx.z * term2_dj.y;
						tdGt_dj.z = t_term1_coeff * dct_dj.z + tx.z * term2_dj.z;
						tdGt_dk.x = t_term1_coeff * dct_dk.x + tx.z * term2_dk.x;
						tdGt_dk.y = t_term1_coeff * dct_dk.y + tx.z * term2_dk.y;
						tdGt_dk.z = t_term1_coeff * dct_dk.z + tx.z * term2_dk.z;

						dG_dj[index_t].x += tdGt_dj.x;
						dG_dj[index_t].y += tdGt_dj.y;
						dG_dj[index_t].z += tdGt_dj.z;

						dG_dkx[m][tid] += tdGt_dk.x;
						dG_dky[m][tid] += tdGt_dk.y;
						dG_dkz[m] += tdGt_dk.z;
					}	// ntsf loop

					int begin_k = (begin_jk + idk) * nsf + npsf;

					if (n == 0 || t_per_atom == 1)																	// updating the dG_dk, values
						for (int m = 0; m < ntsf; m++) {
							index_bm = begin_k + m;
							dGij[index_bm].x += dG_dkx[m][tid];
							dGij[index_bm].y += dG_dky[m][tid];
							dGij[index_bm].z += dG_dkz[m];
						}
					else {
					// if(ii == 2)
					// 	printf("check00... %d %d %d %d %d %d %d %d %f\n", ii, offset, j, k, n, begin_jk, max_nbor_size, begin_k, dG_dj[9].w);
						acc_dGij(dGij, dG_dkx, dG_dky, dG_dkz, begin_k, ntsf, tid, offset, t_per_atom);				// in this case, all threads are processed the same "k"
					// if(ii == 2)
					// 	printf("check22... %d %d %d %d %d %d %d %d %f\n", ii, offset, j, k, n, begin_jk, max_nbor_size, begin_k, dG_dj[9].w);
					}
				}
			}
			int begin_j = (begin_jk + idj) * nsf;
			dGij[begin_j].w = (numtyp)j;
			for (int m = 0; m < nsf; m++) {
				index_bm = begin_j + m;
				dGij[index_bm].x += dG_dj[m].x;
				dGij[index_bm].y += dG_dj[m].y;
				dGij[index_bm].z += dG_dj[m].z;
				//printf("%d, %d, %d, %f�� %f, %f\n", i, m, index_bm, dG_dj[m].x, dG_dj[m].y, dG_dj[m].z);
			}
		}																									// j loop
		acc_Gi(dG_dj, sfnor_scal, sfnor_avg, nsf, tid, t_per_atom, offset);
		//  if(offset == 0)
		//  	for(int n = 0; n < nsf; n++)
		//  		printf("checking....000... %d %d %d %f %f %f %f\n", n, ii, offset, ix.x, ix.y, ix.z, dG_dj[n].w);


		numtyp hidly[10] = { 0.0 };																			// saving the value of hidly layer
		numtyp t_hidly[10] = { 0.0 };
		numtyp hidly_d[10] = { 0.0 };																		// derivate of hidly layer
		numtyp lays_dw[10 * 28] = { 0.0 };																	// for 2 layers multipulity
		numtyp temp_dw[10 * 28] = { 0.0 };
		numtyp hidly_dw[10 * 28] = { 0.0 };																	// using for update		

		//if(offset == 0)
		//	for(int n = 0; n < nsf; n++)
		//		printf("checking... %d %d %f\n", ii, offset, dG_dj[n].w);

		numtyp weight, bias;
		int index_w = 0, index_w2 = 0, index_t;
		int2 nrc_w[3];
		nrc_w[0].x = nnod; nrc_w[0].y = nsf;
		nrc_w[1].x = nnod; nrc_w[1].y = nnod;
		nrc_w[2].x = 1; nrc_w[2].y = nnod;
		for (int n = 0; n < ntl - 1; n++) {
			for (int k = 0; k < nnod; k++) {
				t_hidly[k] = 0.0;
				index_t = k * nsf;
				for (int m = 0; m < nsf; m++) {														// set 0
					lays_dw[index_t + m] = 0.0;
				}
			}
			int actflag = flagact[n];
			for (int k = 0; k < nrc_w[n].x; k++) {
				fetch(bias, k + n * nnod, bias_tex);
				for (int m = 0; m < nrc_w[n].y; m++) {
					//fetch(weight, index_w, weight_tex);
					weight = weight_all[index_w];
					if (n == 0) {
						t_hidly[k] += weight * dG_dj[m].w;
					}
					else {
						t_hidly[k] += weight * hidly[m];
					}
					index_w++;
				}
				t_hidly[k] += bias;
			}
			for (int k = 0; k < nrc_w[n].x; k++) {													// for activation 
				if (actflag == 0) {
					t_hidly[k] = t_hidly[k];
					hidly_d[k] = 1;
				}
				if (actflag == 4) {
					numtyp t_exp = coeff_b * t_hidly[k];
					numtyp t_tanhx = (ucl_exp(t_exp) - ucl_exp(-t_exp)) / (ucl_exp(t_exp) + ucl_exp(-t_exp));
					t_hidly[k] = coeff_a * t_tanhx + coeff_c * t_hidly[k];
					hidly_d[k] = coeff_a * (1.0 - t_tanhx * t_tanhx) * coeff_b + coeff_c;
				}
				hidly[k] = t_hidly[k];
			}
			int index_dw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {													// hidly_d multiply the weight
				for (int m = 0; m < nrc_w[n].y; m++) {
					//fetch(weight, index_w2, weight_tex);											// cannot be used for RTX A5000
					weight = weight_all[index_w2];
					hidly_dw[index_dw] = hidly_d[k] * weight;
					index_w2++;
					index_dw++;
				}
			}																						// for geting dE_dG
			int index_tdw = 0;
			for (int k = 0; k < nrc_w[n].x; k++) {
				for (int m = 0; m < nsf; m++) {
					if (n == 0) {
						temp_dw[k * nsf + m] = hidly_dw[k * nsf + m];
					}
					else {
						for (int j = 0; j < nrc_w[n].y; j++) {
							lays_dw[index_tdw] += hidly_dw[k * nnod + j] * temp_dw[j * nsf + m];
						}
						index_tdw++;
					}
				}
			}
			if (n != 0 && n != ntl - 1)																// updating the temp matrix
				for (int k = 0; k < nrc_w[n].x; k++) {
					for (int m = 0; m < nsf; m++) {
						temp_dw[k * nsf + m] = lays_dw[k * nsf + m];
					}
				}																					// update the tt_dw
		}
		// get energy and force for atom i: wq
		if (offset == 0) {
			engv[ii] = e_scale * hidly[0] + e_shift + e_atom;
			//printf("energy.... %d %f\n", ii, engv[ii]);
		}

		// force for atom j
		acctyp4 temp_f, Fi;
		numtyp scaling;
		Fi.x = 0.0; Fi.y = 0.0; Fi.z = 0.0;
		for (int jj = offset; jj < jnum; jj += t_per_atom) {
			temp_f.x = 0.0; temp_f.y = 0.0; temp_f.z = 0.0;
			int begin_j = (begin_jk + jj) * nsf;
			for (int k = 0; k < nsf; k++) {
				fetch(scaling, k, sfsc_tex);
				temp_f.x -= scaling * lays_dw[k] * dGij[begin_j + k].x * e_scale;
				temp_f.y -= scaling * lays_dw[k] * dGij[begin_j + k].y * e_scale;
				temp_f.z -= scaling * lays_dw[k] * dGij[begin_j + k].z * e_scale;
			}
			Fj[begin_jk + jj].x += temp_f.x;
			Fj[begin_jk + jj].y += temp_f.y;
			Fj[begin_jk + jj].z += temp_f.z;
			Fj[begin_jk + jj].w = dGij[begin_j].w;

			Fi.x -= temp_f.x;
			Fi.y -= temp_f.y;
			Fi.z -= temp_f.z;
		}
		acc_Fi(Fi, force, ii, tid, t_per_atom, offset);
	}	// if ii
}

//----------------------------------------------------------------------
	// updating the force for neighbor
//----------------------------------------------------------------------
__kernel void k_annp_updat(const __global numtyp4* restrict x_,
						   const __global int* restrict newj, 
						   const __global acctyp4* restrict Fj,
						   __global acctyp4* force,
						   __global acctyp2* virial2,
						   __global acctyp4* virial4,
						   const int eflag, const int vflag, 
						   const int begin_i, const int2 gpup) {

	int max_nbor_size = gpup.x;
	int num_atoms = gpup.y;
	int tid = THREAD_ID_X;
	__shared__ int ii;
	__shared__ numtyp4 ix;

	acctyp2 old_v2;
	acctyp4 old_v4;
	for (ii = 0; ii < num_atoms; ) {
		__shared__ acctyp4 tFj[BLOCK_PAIR];
		__shared__ acctyp tvirial[BLOCK_PAIR][6];
		int indexi = ii + begin_i;
		int n_jnum = newj[indexi];
		int begin_jk = ii * max_nbor_size;

		if (tid < n_jnum) {
			int idj = begin_jk + tid;
			int indexj = (int)Fj[idj].w;
			tFj[tid].x = Fj[idj].x;
			tFj[tid].y = Fj[idj].y;
			tFj[tid].z = Fj[idj].z;

			acctyp4 old_f = force[indexj];															// old values
			old_f.w = indexj;
			old_f.x += tFj[tid].x;
			old_f.y += tFj[tid].y;
			old_f.z += tFj[tid].z;
			force[indexj] = old_f;

			if(EVFLAG && vflag) {
				fetch4(ix, indexi, pos_tex);
				numtyp4 jx; fetch4(jx, indexj, pos_tex);
				numtyp delx = ix.x - jx.x;
				numtyp dely = ix.y - jx.y;
				numtyp delz = ix.z - jx.z;	

				tvirial[tid][0] = delx*-tFj[tid].x;
				tvirial[tid][1] = dely*-tFj[tid].y;
				tvirial[tid][2] = delz*-tFj[tid].z;
				tvirial[tid][3] = delx*-tFj[tid].y;
				tvirial[tid][4] = delx*-tFj[tid].z;
				tvirial[tid][5] = dely*-tFj[tid].z;
						
				old_v2 = virial2[indexj];
				old_v4 = virial4[indexj];	
				old_v4.x += 0.5 * tvirial[tid][0];
				old_v4.y += 0.5 * tvirial[tid][1];
				old_v4.z += 0.5 * tvirial[tid][2];
				old_v4.w += 0.5 * tvirial[tid][3];
				old_v2.x += 0.5 * tvirial[tid][4];
				old_v2.y += 0.5 * tvirial[tid][5];
				virial2[indexj] = old_v2;
				virial4[indexj] = old_v4;
			}		
		}
		for (unsigned int s = n_jnum / 2; s > 0; s >>= 1) {
			int idtid = tid + s;
			__syncthreads();
			if (tid < s) {
				if(EVFLAG && vflag) {
					tvirial[tid][0] += tvirial[idtid][0];
					tvirial[tid][1] += tvirial[idtid][1];
					tvirial[tid][2] += tvirial[idtid][2];
					tvirial[tid][3] += tvirial[idtid][3];
					tvirial[tid][4] += tvirial[idtid][4];
					tvirial[tid][5] += tvirial[idtid][5];
				}
			}
			__syncthreads();
			if (s % 2 == 1 && s != 1 && tid == 0) {													// in case of the odd number of value occuring at sub-layers
				idtid -= 1;
				if(EVFLAG && vflag) {
					tvirial[tid][0] += tvirial[idtid][0];
					tvirial[tid][1] += tvirial[idtid][1];
					tvirial[tid][2] += tvirial[idtid][2];
					tvirial[tid][3] += tvirial[idtid][3];
					tvirial[tid][4] += tvirial[idtid][4];
					tvirial[tid][5] += tvirial[idtid][5];
				}
			}
		}
		if (n_jnum % 2 == 1 && tid == 0) {															// for the n_jnum is equal to odd
			int idtid = tid + n_jnum - 1;
			if(EVFLAG && vflag) {
				tvirial[tid][0] += tvirial[idtid][0];
				tvirial[tid][1] += tvirial[idtid][1];
				tvirial[tid][2] += tvirial[idtid][2];
				tvirial[tid][3] += tvirial[idtid][3];
				tvirial[tid][4] += tvirial[idtid][4];
				tvirial[tid][5] += tvirial[idtid][5];
			}
		}
		if (tid == 0) {
			old_v2 = virial2[indexi];
			old_v4 = virial4[indexi];						
			old_v4.x += 0.5 * tvirial[tid][0];
			old_v4.y += 0.5 * tvirial[tid][1];
			old_v4.z += 0.5 * tvirial[tid][2];
			old_v4.w += 0.5 * tvirial[tid][3];
			old_v2.x += 0.5 * tvirial[tid][4];
			old_v2.y += 0.5 * tvirial[tid][5];

			virial2[indexi] = old_v2;
			virial4[indexi] = old_v4;
		}
		__syncthreads();		
		ii++;
	}
}
